#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <set>
#include <chrono>
#include <string>


#define BOARD_SIZE 81
#define SUB_BOARD_SIZE 9
#define SUB_BOARD_DIM 3

#define THREAD_PER_BLOCK 1024

#pragma region Boards
/*Boards*/
// https://www.puzzles.ca/sudoku_puzzles/sudoku_easy_487.html
int* test_board_easy = new int[81]{ 0, 7, 0, 0, 0, 1, 0, 0, 0,
									6, 0, 0, 0, 0, 0, 0, 0, 0,
									0, 0, 0, 0, 5, 3, 0, 0, 0,
									0, 0, 0, 8, 0, 0, 0, 2, 0,
									0, 3, 0, 0, 4, 7, 1, 6, 0,
									4, 0, 0, 0, 0, 0, 5, 7, 0,
									0, 0, 0, 0, 1, 0, 7, 5, 0,
									0, 6, 0, 5, 2, 0, 0, 4, 0,
									3, 0, 0, 0, 0, 9, 0, 8, 6 };

int* easy_test_answer = new int[81]{ 5, 7, 9, 6, 8, 1, 4, 3, 2,
									6, 2, 3, 7, 9, 4, 8, 1, 5,
									1, 8, 4, 2, 5, 3, 6, 9, 7,
									7, 1, 6, 8, 3, 5, 9, 2, 4,
									2, 3, 5, 9, 4, 7, 1, 6, 8,
									4, 9, 8, 1, 6, 2, 5, 7, 3,
									8, 4, 2, 3, 1, 6, 7, 5, 9,
									9, 6, 7, 5, 2, 8, 3, 4, 1,
									3, 5, 1, 4, 7, 9, 2, 8, 6 };

int* test_board_easy2 = new int[81]{ 0, 0, 0, 0, 9, 0, 0, 0, 0,
									  0, 3, 0, 0, 0, 0, 7, 4, 2,
									  6, 1, 0, 0, 3, 0, 0, 0, 0,
									  0, 0, 8, 0, 0, 0, 0, 0, 0,
									  0, 2, 9, 1, 4, 7, 0, 0, 8,
									  4, 5, 0, 0, 0, 0, 0, 7, 0,
									  0, 4, 0, 0, 0, 0, 0, 6, 0,
									  5, 0, 0, 6, 0, 0, 0, 3, 0,
									  0, 0, 0, 0, 8, 1, 2, 0, 0 };

int* easy_test2_answer = new int[81]{ 2, 8, 4, 7, 9, 5, 3, 1, 6,
									   9, 3, 5, 8, 1, 6, 7, 4, 2,
									   6, 1, 7, 2, 3, 4, 5, 8, 9,
									   1, 7, 8, 5, 6, 9, 4, 2, 3,
									   3, 2, 9, 1, 4, 7, 6, 5, 8,
									   4, 5, 6, 3, 2, 8, 9, 7, 1,
									   8, 4, 2, 9, 5, 3, 1, 6, 7,
									   5, 9, 1, 6, 7, 2, 8, 3, 4,
									   7, 6, 3, 4, 8, 1, 2, 9, 5 };

// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_medium = new int[81]{ 0, 0, 0, 0, 0, 0, 3, 0, 0,
									   8, 5, 2, 3, 0, 0, 0, 0, 1,
									   0, 9, 0, 2, 0, 0, 0, 0, 4,
									   9, 7, 4, 0, 0, 0, 0, 0, 0,
									   0, 1, 0, 0, 6, 0, 0, 0, 0,
									   0, 0, 0, 0, 4, 0, 0, 0, 0,
									   6, 0, 9, 0, 8, 0, 0, 3, 7,
									   3, 0, 0, 0, 0, 0, 0, 6, 0,
									   0, 2, 0, 0, 0, 5, 0, 0, 0 };

int* medium_test_answer = new int[81]{ 4, 6, 7, 9, 1, 8, 3, 2, 5,
										8, 5, 2, 3, 7, 4, 6 ,9, 1,
										1, 9, 3, 2, 5, 6, 7, 8, 4,
										9, 7, 4, 5, 2, 3, 8, 1, 6,
										2, 1, 8, 7, 6, 9, 4, 5, 3,
										5, 3, 6, 8, 4, 1, 2, 7, 9,
										6, 4, 9, 1, 8, 2, 5, 3, 7,
										3, 8, 5, 4, 9, 7, 1, 6, 2,
										7, 2, 1, 6, 3, 5, 9, 4, 8 };


// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_hard = new int[81]{ 0, 7, 0, 5, 0, 6, 0, 0, 0,
									 4, 0, 3, 0, 0, 0, 0, 0, 1,
									 0, 6, 0, 0, 0, 0, 9, 0, 7,
									 0, 0, 0, 7, 3, 0, 8, 2, 0,
									 8, 0, 4, 0, 5, 0, 0, 7, 3,
									 0, 9, 0, 0, 2, 0, 0, 0, 5,
									 0, 0, 1, 0, 0, 0, 0, 0, 0,
									 0, 0, 0, 1, 0, 0, 2, 0, 6,
									 0, 0, 0, 3, 8, 2, 0, 0, 0 };

//int* hard_test_answer = new int[81]{    4, 6, 7, 9, 1, 8, 3, 2, 5,
//										8, 5, 2, 3, 7, 4, 6 ,9, 1,
//										1, 9, 3, 2, 5, 6, 7, 8, 4,
//										9, 7, 4, 5, 2, 3, 8, 1, 6,
//										2, 1, 8, 7, 6, 9, 4, 5, 3,
//										5, 3, 6, 8, 4, 1, 2, 7, 9,
//										6, 4, 9, 1, 8, 2, 5, 3, 7,
//										3, 8, 5, 4, 9, 7, 1, 6, 2,
//										7, 2, 1, 6, 3, 5, 9, 4, 8 };

// http://www.ams.org/notices/200904/rtx090400460p.pdf
int* test_board_diabolical = new int[81]{ 0, 9, 0, 7, 0, 0, 8, 6, 0,
										  0, 3, 1, 0, 0, 5, 0, 2, 0,
										  8, 0, 6, 0, 0, 0, 0, 0, 0,
										  0, 0, 7, 0, 5, 0, 0, 0, 6,
										  0, 0, 0, 3, 0, 7, 0, 0, 0,
										  5, 0, 0, 0, 1, 0, 7, 0, 0,
										  0, 0, 0, 0, 0, 0, 1, 0, 9,
										  0, 2, 0, 6, 0, 0, 3, 5, 0,
										  0, 5, 4, 0, 0, 8, 0, 7, 0 };

int* diabolical_test_answer = new int[81]{ 2, 9, 5, 7, 4, 3, 8, 6, 1,
										   4, 3, 1, 8, 6, 5, 9, 2, 7,
										   8, 7, 6, 1, 9, 2, 5, 4, 3,
										   3, 8, 7, 4, 5, 9, 2, 1, 6,
										   6, 1, 2, 3, 8, 7, 4, 9, 5,
										   5, 4, 9, 2, 1, 6, 7, 3, 8,
										   7, 6, 3, 5, 2, 4, 1, 8, 9,
										   9, 2, 8, 6, 7, 1, 3, 5, 4,
										   1, 5, 4, 9, 3, 8, 6, 7, 2 };

#pragma endregion


__device__ bool row_check_dev(const int* _board, int _board_root, int _row, int _entry, int loc)
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i] == _entry)
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check_dev(const int* _board, int _board_root, int _col, int _entry, int loc)
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check_dev(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry_dev(const int* _board, int _board_root, int _row, int _col, int _entry, int loc)
{
	return row_check_dev(_board, _board_root, _row, _entry, loc) &&
		column_check_dev(_board, _board_root, _col, _entry, loc) &&
		grid_check_dev(_board, _board_root, _row, _col, _entry, loc);
}

// Returns whether or not it is valid to put a value in specified location for this board
__device__ bool IsLegal(int *_board, int _loc, int _val)
{
	if (is_legal_entry_dev(_board, SUB_BOARD_SIZE, _loc / SUB_BOARD_DIM, _loc % SUB_BOARD_DIM, _val, _loc)) {
		_board[_loc] = _val;
		return true;
	}

	return false;
}

// Find next empty cell in passed in board
__device__ int FindNextEmptyCell(int* board) {
	for (int i = 0; i < BOARD_SIZE; i++) {
		if (board[i] == 0) {
			return i;
		}
	}
	return -1;
}

// new boards points to the end of the filled in prev boards
__global__ void GenerateBoardsByCell(int *prev_boards, int prev_board_num, int *new_boards, int *new_board_num) {

	// gives the previous board number to look at
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	// each thread will look at 1 previous board 
	// thread only does work if the amount of previous boards greater than its thread num
	// maybe should use a for loop in the case a thread has to do more than one thread. Will this ever occur?
	if (t_idx < prev_board_num) {
		int prev_board_start = t_idx * BOARD_SIZE;
		int* thread_prev_board = (int*)malloc(sizeof(int) * BOARD_SIZE);

		for (int i = 0; i < SUB_BOARD_SIZE; i++) { // read prev board into a sudoku sized local array
			thread_prev_board[i] = prev_boards[prev_board_start + i];
		}

		// find next index we can add to
		int empty_cell = FindNextEmptyCell(thread_prev_board);
		if (empty_cell == -1) { // Board is full
			return;
		}

		// Now try all possible numbers in this cell that islegal
		for (int i = 1; i <= 9; i++) {
			if (IsLegal(thread_prev_board, empty_cell, i)) { // number can go in this cell

				// where to start appending for the new board
				int new_board_offset = atomicAdd(&new_board_num, 1); // increment amount of boards at the new depth

				for (int j = 0; j < BOARD_SIZE; j++) {
					int ind = (new_board_offset * BOARD_SIZE) + j;

					new_boards[ind] = thread_prev_board[j];
				}
			}
		}
	}

}

__global__ void SolveBoard(int **_all_boards, int *_solved_board) {

}

void solve(int *board, int depth) {

	int h_solution[BOARD_SIZE];

	int *new_boards;
	int *old_boards;
	int *solution;
	int *next_board_num;

	const int memSize = 81 * pow(9, depth);

	// allocate device memory and set everything to 0
	hipMalloc(&next_board_num, sizeof(int));
	hipMalloc(&solution, BOARD_SIZE * sizeof(int));
	hipMalloc(&new_boards, memSize * sizeof(int));
	hipMalloc(&old_boards, memSize * sizeof(int));

	hipMemset(next_board_num, 0, sizeof(int));
	hipMemset(solution, 0, BOARD_SIZE * sizeof(int));
	hipMemset(new_boards, 0, memSize * sizeof(int));
	hipMemset(old_boards, 0, memSize * sizeof(int));


	// BFS on GPU

	// copy starting board to device memory
	hipMemcpy(old_boards, board, BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice);

	// 1 due to starting board
	int prev_board_num = 1;
	int *prev_boards = new_boards;
	for (int i = 0; i < depth; i++) {

		// Need 1 thread per 
		int num_blocks = (prev_board_num + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

		// need to set next board num to 0 before we start next depth
		hipMemset(next_board_num, 0, sizeof(int));

		GenerateBoardsByCell << <num_blocks, THREAD_PER_BLOCK >> > (prev_boards, prev_board_num, new_boards, next_board_num);

		int* temp_board = prev_boards;
		prev_boards = new_boards;
		new_boards = temp_board;

		// update the amount of boards in previous depth for next iteration
		hipMemcpy(&prev_board_num, next_board_num, sizeof(int), hipMemcpyDeviceToHost);
	}


	// DFS on GPU
	// Working on it...
}

int main()
{
	// Solve medium sized board and go to depth 7 of BFS
	solve(test_board_medium, 7);
}
