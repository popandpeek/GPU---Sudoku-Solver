#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <stdlib.h>
#include <cmath>
#include <vector>
#include <cstring>
#include <iostream>
#include <fstream>

#define THREADS_PER_BLOCK 512
#define BOARD_SIZE 81
#define SUB_BOARD_SIZE 9
#define SUB_BOARD_DIM 3

void print_board_1d(int *board) {

	char* border = new char[26]{ "|-------+-------+-------|" };

	std::cout << border << std::endl;
	int split = sqrt(9);
	for (int i = 0; i < 9 * 9; i++) {
		if (i % 9 == 0) {
			std::cout << "| ";
		}
		else if (i % split == 0) {
			std::cout << "| ";
		}

		int value = board[i];
		if (value != 0) {
			std::cout << value << " ";
		}
		else {
			std::cout << ". ";
		}

		if (i % 9 == 9 - 1) {
			std::cout << "|" << std::endl;

			if (((i + 1) % (9 * 9 / split)) == 0) {
				std::cout << border << std::endl;
			}
		}
	}
	std::cout << std::endl;
}


#pragma region Boards
/*Boards*/
// https://www.puzzles.ca/sudoku_puzzles/sudoku_easy_487.html
int* test_board_easy = new int[81]{ 0, 7, 0, 0, 0, 1, 0, 0, 0,
									6, 0, 0, 0, 0, 0, 0, 0, 0,
									0, 0, 0, 0, 5, 3, 0, 0, 0,
									0, 0, 0, 8, 0, 0, 0, 2, 0,
									0, 3, 0, 0, 4, 7, 1, 6, 0,
									4, 0, 0, 0, 0, 0, 5, 7, 0,
									0, 0, 0, 0, 1, 0, 7, 5, 0,
									0, 6, 0, 5, 2, 0, 0, 4, 0,
									3, 0, 0, 0, 0, 9, 0, 8, 6 };

int* easy_test_answer = new int[81]{ 5, 7, 9, 6, 8, 1, 4, 3, 2,
									6, 2, 3, 7, 9, 4, 8, 1, 5,
									1, 8, 4, 2, 5, 3, 6, 9, 7,
									7, 1, 6, 8, 3, 5, 9, 2, 4,
									2, 3, 5, 9, 4, 7, 1, 6, 8,
									4, 9, 8, 1, 6, 2, 5, 7, 3,
									8, 4, 2, 3, 1, 6, 7, 5, 9,
									9, 6, 7, 5, 2, 8, 3, 4, 1,
									3, 5, 1, 4, 7, 9, 2, 8, 6 };

int* test_board_easy2 = new int[81]{ 0, 0, 0, 0, 9, 0, 0, 0, 0,
									  0, 3, 0, 0, 0, 0, 7, 4, 2,
									  6, 1, 0, 0, 3, 0, 0, 0, 0,
									  0, 0, 8, 0, 0, 0, 0, 0, 0,
									  0, 2, 9, 1, 4, 7, 0, 0, 8,
									  4, 5, 0, 0, 0, 0, 0, 7, 0,
									  0, 4, 0, 0, 0, 0, 0, 6, 0,
									  5, 0, 0, 6, 0, 0, 0, 3, 0,
									  0, 0, 0, 0, 8, 1, 2, 0, 0 };

int* easy_test2_answer = new int[81]{ 2, 8, 4, 7, 9, 5, 3, 1, 6,
									   9, 3, 5, 8, 1, 6, 7, 4, 2,
									   6, 1, 7, 2, 3, 4, 5, 8, 9,
									   1, 7, 8, 5, 6, 9, 4, 2, 3,
									   3, 2, 9, 1, 4, 7, 6, 5, 8,
									   4, 5, 6, 3, 2, 8, 9, 7, 1,
									   8, 4, 2, 9, 5, 3, 1, 6, 7,
									   5, 9, 1, 6, 7, 2, 8, 3, 4,
									   7, 6, 3, 4, 8, 1, 2, 9, 5 };

// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_medium = new int[81]{ 0, 0, 0, 0, 0, 0, 3, 0, 0,
									   8, 5, 2, 3, 0, 0, 0, 0, 1,
									   0, 9, 0, 2, 0, 0, 0, 0, 4,
									   9, 7, 4, 0, 0, 0, 0, 0, 0,
									   0, 1, 0, 0, 6, 0, 0, 0, 0,
									   0, 0, 0, 0, 4, 0, 0, 0, 0,
									   6, 0, 9, 0, 8, 0, 0, 3, 7,
									   3, 0, 0, 0, 0, 0, 0, 6, 0,
									   0, 2, 0, 0, 0, 5, 0, 0, 0 };

int* medium_test_answer = new int[81]{ 4, 6, 7, 9, 1, 8, 3, 2, 5,
										8, 5, 2, 3, 7, 4, 6 ,9, 1,
										1, 9, 3, 2, 5, 6, 7, 8, 4,
										9, 7, 4, 5, 2, 3, 8, 1, 6,
										2, 1, 8, 7, 6, 9, 4, 5, 3,
										5, 3, 6, 8, 4, 1, 2, 7, 9,
										6, 4, 9, 1, 8, 2, 5, 3, 7,
										3, 8, 5, 4, 9, 7, 1, 6, 2,
										7, 2, 1, 6, 3, 5, 9, 4, 8 };


// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_hard = new int[81]{ 0, 7, 0, 5, 0, 6, 0, 0, 0,
									 4, 0, 3, 0, 0, 0, 0, 0, 1,
									 0, 6, 0, 0, 0, 0, 9, 0, 7,
									 0, 0, 0, 7, 3, 0, 8, 2, 0,
									 8, 0, 4, 0, 5, 0, 0, 7, 3,
									 0, 9, 0, 0, 2, 0, 0, 0, 5,
									 0, 0, 1, 0, 0, 0, 0, 0, 0,
									 0, 0, 0, 1, 0, 0, 2, 0, 6,
									 0, 0, 0, 3, 8, 2, 0, 0, 0 };

//int* hard_test_answer = new int[81]{    4, 6, 7, 9, 1, 8, 3, 2, 5,
//										8, 5, 2, 3, 7, 4, 6 ,9, 1,
//										1, 9, 3, 2, 5, 6, 7, 8, 4,
//										9, 7, 4, 5, 2, 3, 8, 1, 6,
//										2, 1, 8, 7, 6, 9, 4, 5, 3,
//										5, 3, 6, 8, 4, 1, 2, 7, 9,
//										6, 4, 9, 1, 8, 2, 5, 3, 7,
//										3, 8, 5, 4, 9, 7, 1, 6, 2,
//										7, 2, 1, 6, 3, 5, 9, 4, 8 };

// http://www.ams.org/notices/200904/rtx090400460p.pdf
int* test_board_diabolical = new int[81]{ 0, 9, 0, 7, 0, 0, 8, 6, 0,
										  0, 3, 1, 0, 0, 5, 0, 2, 0,
										  8, 0, 6, 0, 0, 0, 0, 0, 0,
										  0, 0, 7, 0, 5, 0, 0, 0, 6,
										  0, 0, 0, 3, 0, 7, 0, 0, 0,
										  5, 0, 0, 0, 1, 0, 7, 0, 0,
										  0, 0, 0, 0, 0, 0, 1, 0, 9,
										  0, 2, 0, 6, 0, 0, 3, 5, 0,
										  0, 5, 4, 0, 0, 8, 0, 7, 0 };

int* diabolical_test_answer = new int[81]{ 2, 9, 5, 7, 4, 3, 8, 6, 1,
										   4, 3, 1, 8, 6, 5, 9, 2, 7,
										   8, 7, 6, 1, 9, 2, 5, 4, 3,
										   3, 8, 7, 4, 5, 9, 2, 1, 6,
										   6, 1, 2, 3, 8, 7, 4, 9, 5,
										   5, 4, 9, 2, 1, 6, 7, 3, 8,
										   7, 6, 3, 5, 2, 4, 1, 8, 9,
										   9, 2, 8, 6, 7, 1, 3, 5, 4,
										   1, 5, 4, 9, 3, 8, 6, 7, 2 };

#pragma endregion

// function to examine if there are conflicts or not if cell [row][col] is num
__device__
bool noConflicts(int matrix[BOARD_SIZE], int row, int col, int num) {
	if (num <= 0 || num > SUB_BOARD_SIZE) return false;
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		if (i == row)   continue;
		if (matrix[i * SUB_BOARD_SIZE + col] == num) {
			return false;
		}
	}

	for (int j = 0; j < SUB_BOARD_SIZE; j++) {
		if (j == col)   continue;
		if (matrix[row * SUB_BOARD_SIZE + j] == num) {
			return false;
		}
	}

	for (int i = 0; i < SUB_BOARD_DIM; i++) {
		for (int j = 0; j < SUB_BOARD_DIM; j++) {
			int mat_row = (row / SUB_BOARD_DIM)*SUB_BOARD_DIM + i;
			int mat_col = (col / SUB_BOARD_DIM)*SUB_BOARD_DIM + j;
			if (mat_row == row && mat_col == col)   continue;
			if (matrix[mat_row * SUB_BOARD_SIZE + mat_col] == num) {
				return false;
			}
		}
	}
	return true;
}

__device__ bool row_check_dev(const int* _board, int _board_root, int _row, int _entry, int loc)
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i] == _entry)
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check_dev(const int* _board, int _board_root, int _col, int _entry, int loc)
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check_dev(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry_dev(const int* _board, int _board_root, int _row, int _col, int _entry, int loc)
{
	return row_check_dev(_board, _board_root, _row, _entry, loc) &&
		column_check_dev(_board, _board_root, _col, _entry, loc) &&
		grid_check_dev(_board, _board_root, _row, _col, _entry, loc);
}

// Returns whether or not it is valid to put a value in specified location for this board
__device__ bool IsLegal(int *_board, int _loc, int _val)
{
	if (is_legal_entry_dev(_board, SUB_BOARD_SIZE, _loc / SUB_BOARD_DIM, _loc % SUB_BOARD_DIM, _val, _loc)) {
		_board[_loc] = _val;
		return true;
	}

	return false;
}



__device__ bool row_check_dev(const int* _board, int _board_root, int _row, int _entry, int loc, int _boardStart)
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i + _boardStart] == _entry)
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check_dev(const int* _board, int _board_root, int _col, int _entry, int loc, int _boardStart)
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i + _boardStart] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check_dev(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc, int _boardStart)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind + _boardStart] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry_dev(const int* _board, int _board_root, int _row, int _col, int _entry, int loc, int _boardStart)
{
	return row_check_dev(_board, _board_root, _row, _entry, loc, _boardStart) &&
		column_check_dev(_board, _board_root, _col, _entry, loc, _boardStart) &&
		grid_check_dev(_board, _board_root, _row, _col, _entry, loc, _boardStart);
}

// Returns whether or not it is valid to put a value in specified location for this board
__device__ bool IsLegal(int *_board, int _loc, int _val, int _boardStart)
{
	if (is_legal_entry_dev(_board, SUB_BOARD_SIZE, _loc / SUB_BOARD_DIM + _boardStart, _loc % SUB_BOARD_DIM + _boardStart, _val, _loc, _boardStart)) {
		_board[_loc] = _val;
		return true;
	}

	return false;
}

// Find next empty cell in passed in board
__device__ int FindNextEmptyCell(int* board, int _boardStart) {
	for (int i = 0; i < BOARD_SIZE; i++)
	{
		if (board[i + _boardStart] == 0) {
			return i;
		}
	}
	return -1;
}

// new boards points to the end of the filled in prev boards
__global__ void GenerateBoardsByCell(int *old_boards, int old_board_num, int *new_boards, int *new_board_num) {

	// gives the previous board number to look at
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	// each thread will look at 1 previous board 
	// thread only does work if the amount of previous boards greater than its thread num
	// maybe should use a for loop in the case a thread has to do more than one thread. Will this ever occur?
	if (t_idx < old_board_num) {
		int old_board_start = t_idx * BOARD_SIZE;


		// find next index we can add to
		int empty_cell_ind = FindNextEmptyCell(old_boards, old_board_start);
		if (empty_cell_ind == -1) { // Board is full
			return;
		}

		// Now try all possible numbers in this cell that is legal
		for (int i = 1; i <= 9; i++) {
			if (IsLegal(old_boards, empty_cell_ind, i, old_board_start)) { // number can go in this cell

				// where to start appending for the new board
				int new_board_offset = atomicAdd(new_board_num, 1); // increment amount of boards at the new depth

				for (int j = 0; j < BOARD_SIZE; j++)
				{
					int ind = (new_board_offset * BOARD_SIZE) + j;

					new_boards[ind] = old_boards[j + old_board_start];
				}
			}
		}
	}

}

// Use DFS to solve specified board per thread
__global__ void SolveBoard(int *boards, int total_boards, int* solution) {
	int t_idx = blockIdx.x * blockDim.x + threadIdx.x;


	if (t_idx < total_boards) {

		int empty_cells = 0;

		int* empty_indices = (int*)malloc(sizeof(int) * BOARD_SIZE);
		int* thread_board = (int*)malloc(sizeof(int) * BOARD_SIZE);

		int board_start = t_idx * BOARD_SIZE;
		for (int i = 0; i < BOARD_SIZE; i++) {
			int ind = board_start + i;

			thread_board[i] = boards[ind];
			if (thread_board[i] == 0) {
				empty_indices[empty_cells] = i;
				empty_cells++;
			}
		}

		int filled_empty_cells = 0;
		while (filled_empty_cells >= 0 && filled_empty_cells < empty_cells) {

			int next_cell = empty_indices[filled_empty_cells];
			int row = next_cell / SUB_BOARD_SIZE;
			int col = next_cell % SUB_BOARD_SIZE;

			int val = thread_board[next_cell] + 1;
			thread_board[next_cell]++;

			if (noConflicts(thread_board, row, col, val)) { // IsLegal does not work here?
				filled_empty_cells++;
			}
			else if (thread_board[next_cell] >= SUB_BOARD_SIZE) {
				thread_board[next_cell] = 0;
				filled_empty_cells--;
			}
		}
		if (filled_empty_cells == empty_cells) {
			memcpy(solution, thread_board, BOARD_SIZE * sizeof(int));
		}
	}
}

// Every additional depth guesses one cell with every possible potential
void GenerateBoardsBFS(int* prev_boards, int* new_board_num, int* new_boards, int depth) {

	// 1 because of the first board. This will then change iteration according to the permutations at each depth
	int prev_board_num = 1;

	for (int i = 0; i < depth; i++) {
		int block_num = (prev_board_num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		hipMemset(new_board_num, 0, sizeof(int));
		GenerateBoardsByCell << <block_num, THREADS_PER_BLOCK >> > (prev_boards, prev_board_num, new_boards, new_board_num);
		int* tmp = prev_boards;
		prev_boards = new_boards;
		new_boards = tmp;
		hipMemcpy(&prev_board_num, new_board_num, sizeof(int), hipMemcpyDeviceToHost);
	}
}

// The main solve function
void solve_board(int * board, int depth) {

	// Board that wil FIRST hold the old depth boards (first board)
	int *old_boards;

	// This needs to be a pointer since we want to increment in a device function
	int *old_board_num;

	// Clear storage until second iteration
	int *new_boards;

	// Solution
	int *solution;
	int h_solution[BOARD_SIZE];
	memset(h_solution, 0, BOARD_SIZE * sizeof(int));

	// Theoretical upper bound of boards each cell of a level having all 9 potentials
	const int memSize = 81 * pow(9, depth);

	// alloc device memory
	hipMalloc(&old_board_num, sizeof(int));
	hipMalloc(&new_boards, memSize * sizeof(int));
	hipMalloc(&old_boards, memSize * sizeof(int));
	hipMalloc(&solution, BOARD_SIZE * sizeof(int));


	hipMemset(old_board_num, 0, sizeof(int));
	hipMemset(old_boards, 0, memSize * sizeof(int));
	hipMemset(new_boards, 0, memSize * sizeof(int));
	hipMemset(solution, 0, BOARD_SIZE * sizeof(int));

	// Copy the starting board into our storage array 
	hipMemcpy(old_boards, board, BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice);

	// generates a set of boards with the first depth cells filled in
	GenerateBoardsBFS(old_boards, old_board_num, new_boards, depth);

	// get the total number of boards back
	int total_board_num = 1;
	hipMemcpy(&total_board_num, old_board_num, sizeof(int), hipMemcpyDeviceToHost);

	// Now we solve each board per thread on the GPU by DFS
	int block_num = (total_board_num + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	SolveBoard << <total_board_num, THREADS_PER_BLOCK >> > (old_boards, total_board_num, solution);
	hipDeviceSynchronize();

	hipMemcpy(h_solution, solution, BOARD_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	// Get the solution and print for correctness
	print_board_1d(h_solution);


	// free all used memory
	hipFree(new_boards);
	hipFree(&old_boards);
	hipFree(&old_board_num);
	hipFree(&solution);
}


int main(int argc, char* argv[]) {

	int* board = new int[81]{		  
		0, 0, 0, 0, 0, 0, 3, 0, 0,
		8, 5, 2, 3, 0, 0, 0, 0, 1,
		0, 9, 0, 2, 0, 0, 0, 0, 4,
		9, 7, 4, 0, 0, 0, 0, 0, 0,
		0, 1, 0, 0, 6, 0, 0, 0, 0,
		0, 0, 0, 0, 4, 0, 0, 0, 0,
		6, 0, 9, 0, 8, 0, 0, 3, 7,
		3, 0, 0, 0, 0, 0, 0, 6, 0,
		0, 2, 0, 0, 0, 5, 0, 0, 0 };

	solve_board(board, 5);

	return 0;
}