#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <set>
#include <chrono>
#include <string>


#define BOARD_SIZE 81
#define SUB_BOARD_SIZE 9
#define SUB_BOARD_DIM 3

#pragma region Boards
/*Boards*/
// https://www.puzzles.ca/sudoku_puzzles/sudoku_easy_487.html
int* test_board_easy = new int[81]{ 0, 7, 0, 0, 0, 1, 0, 0, 0,
									6, 0, 0, 0, 0, 0, 0, 0, 0,
									0, 0, 0, 0, 5, 3, 0, 0, 0,
									0, 0, 0, 8, 0, 0, 0, 2, 0,
									0, 3, 0, 0, 4, 7, 1, 6, 0,
									4, 0, 0, 0, 0, 0, 5, 7, 0,
									0, 0, 0, 0, 1, 0, 7, 5, 0,
									0, 6, 0, 5, 2, 0, 0, 4, 0,
									3, 0, 0, 0, 0, 9, 0, 8, 6 };

int* easy_test_answer = new int[81]{ 5, 7, 9, 6, 8, 1, 4, 3, 2,
									6, 2, 3, 7, 9, 4, 8, 1, 5,
									1, 8, 4, 2, 5, 3, 6, 9, 7,
									7, 1, 6, 8, 3, 5, 9, 2, 4,
									2, 3, 5, 9, 4, 7, 1, 6, 8,
									4, 9, 8, 1, 6, 2, 5, 7, 3,
									8, 4, 2, 3, 1, 6, 7, 5, 9,
									9, 6, 7, 5, 2, 8, 3, 4, 1,
									3, 5, 1, 4, 7, 9, 2, 8, 6 };

int* test_board_easy2 = new int[81]{ 0, 0, 0, 0, 9, 0, 0, 0, 0,
									  0, 3, 0, 0, 0, 0, 7, 4, 2,
									  6, 1, 0, 0, 3, 0, 0, 0, 0,
									  0, 0, 8, 0, 0, 0, 0, 0, 0,
									  0, 2, 9, 1, 4, 7, 0, 0, 8,
									  4, 5, 0, 0, 0, 0, 0, 7, 0,
									  0, 4, 0, 0, 0, 0, 0, 6, 0,
									  5, 0, 0, 6, 0, 0, 0, 3, 0,
									  0, 0, 0, 0, 8, 1, 2, 0, 0 };

int* easy_test2_answer = new int[81]{ 2, 8, 4, 7, 9, 5, 3, 1, 6,
									   9, 3, 5, 8, 1, 6, 7, 4, 2,
									   6, 1, 7, 2, 3, 4, 5, 8, 9,
									   1, 7, 8, 5, 6, 9, 4, 2, 3,
									   3, 2, 9, 1, 4, 7, 6, 5, 8,
									   4, 5, 6, 3, 2, 8, 9, 7, 1,
									   8, 4, 2, 9, 5, 3, 1, 6, 7,
									   5, 9, 1, 6, 7, 2, 8, 3, 4,
									   7, 6, 3, 4, 8, 1, 2, 9, 5 };

// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_medium = new int[81]{ 0, 0, 0, 0, 0, 0, 3, 0, 0,
									   8, 5, 2, 3, 0, 0, 0, 0, 1,
									   0, 9, 0, 2, 0, 0, 0, 0, 4,
									   9, 7, 4, 0, 0, 0, 0, 0, 0,
									   0, 1, 0, 0, 6, 0, 0, 0, 0,
									   0, 0, 0, 0, 4, 0, 0, 0, 0,
									   6, 0, 9, 0, 8, 0, 0, 3, 7,
									   3, 0, 0, 0, 0, 0, 0, 6, 0,
									   0, 2, 0, 0, 0, 5, 0, 0, 0 };

int* medium_test_answer = new int[81]{ 4, 6, 7, 9, 1, 8, 3, 2, 5,
										8, 5, 2, 3, 7, 4, 6 ,9, 1,
										1, 9, 3, 2, 5, 6, 7, 8, 4,
										9, 7, 4, 5, 2, 3, 8, 1, 6,
										2, 1, 8, 7, 6, 9, 4, 5, 3,
										5, 3, 6, 8, 4, 1, 2, 7, 9,
										6, 4, 9, 1, 8, 2, 5, 3, 7,
										3, 8, 5, 4, 9, 7, 1, 6, 2,
										7, 2, 1, 6, 3, 5, 9, 4, 8 };


// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_hard = new int[81]{ 0, 7, 0, 5, 0, 6, 0, 0, 0,
									 4, 0, 3, 0, 0, 0, 0, 0, 1,
									 0, 6, 0, 0, 0, 0, 9, 0, 7,
									 0, 0, 0, 7, 3, 0, 8, 2, 0,
									 8, 0, 4, 0, 5, 0, 0, 7, 3,
									 0, 9, 0, 0, 2, 0, 0, 0, 5,
									 0, 0, 1, 0, 0, 0, 0, 0, 0,
									 0, 0, 0, 1, 0, 0, 2, 0, 6,
									 0, 0, 0, 3, 8, 2, 0, 0, 0 };

//int* hard_test_answer = new int[81]{    4, 6, 7, 9, 1, 8, 3, 2, 5,
//										8, 5, 2, 3, 7, 4, 6 ,9, 1,
//										1, 9, 3, 2, 5, 6, 7, 8, 4,
//										9, 7, 4, 5, 2, 3, 8, 1, 6,
//										2, 1, 8, 7, 6, 9, 4, 5, 3,
//										5, 3, 6, 8, 4, 1, 2, 7, 9,
//										6, 4, 9, 1, 8, 2, 5, 3, 7,
//										3, 8, 5, 4, 9, 7, 1, 6, 2,
//										7, 2, 1, 6, 3, 5, 9, 4, 8 };

// http://www.ams.org/notices/200904/rtx090400460p.pdf
int* test_board_diabolical = new int[81]{ 0, 9, 0, 7, 0, 0, 8, 6, 0,
										  0, 3, 1, 0, 0, 5, 0, 2, 0,
										  8, 0, 6, 0, 0, 0, 0, 0, 0,
										  0, 0, 7, 0, 5, 0, 0, 0, 6,
										  0, 0, 0, 3, 0, 7, 0, 0, 0,
										  5, 0, 0, 0, 1, 0, 7, 0, 0,
										  0, 0, 0, 0, 0, 0, 1, 0, 9,
										  0, 2, 0, 6, 0, 0, 3, 5, 0,
										  0, 5, 4, 0, 0, 8, 0, 7, 0 };

int* diabolical_test_answer = new int[81]{ 2, 9, 5, 7, 4, 3, 8, 6, 1,
										   4, 3, 1, 8, 6, 5, 9, 2, 7,
										   8, 7, 6, 1, 9, 2, 5, 4, 3,
										   3, 8, 7, 4, 5, 9, 2, 1, 6,
										   6, 1, 2, 3, 8, 7, 4, 9, 5,
										   5, 4, 9, 2, 1, 6, 7, 3, 8,
										   7, 6, 3, 5, 2, 4, 1, 8, 9,
										   9, 2, 8, 6, 7, 1, 3, 5, 4,
										   1, 5, 4, 9, 3, 8, 6, 7, 2 };

#pragma endregion


__device__ bool row_check_dev(const int* _board, int _board_root, int _row, int _entry, int loc)
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i] == _entry)
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check_dev(const int* _board, int _board_root, int _col, int _entry, int loc)
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check_dev(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry_dev(const int* _board, int _board_root, int _row, int _col, int _entry, int loc)
{
	return row_check_dev(_board, _board_root, _row, _entry, loc) &&
		column_check_dev(_board, _board_root, _col, _entry, loc) &&
		grid_check_dev(_board, _board_root, _row, _col, _entry, loc);
}

// Returns whether or not it is valid to put a value in specified location for this board
__device__ bool IsLegal(int *_board, int _loc, int _val)
{
	if (is_legal_entry_dev(_board, SUB_BOARD_SIZE, _loc / SUB_BOARD_DIM, _loc % SUB_BOARD_DIM, _val, _loc)) {
		_board[_loc] = _val;
		return true;
	}

	return false;
}

// Find next empty cell in passed in board
__device__ int FindNextEmptyCell(int* board) {
	for (int i = 0; i < BOARD_SIZE; i++) {
		if (board[i] == 0) {
			return i;
		}
	}
	return -1;
}

// new boards points to the end of the filled in prev boards
__global__ void GenerateBoardsByCell(int *prev_boards, int prev_board_num, int *new_boards, int *new_board_num) {

	// gives the previous board number to look at
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	// each thread will look at 1 previous board 
	// thread only does work if the amount of previous boards greater than its thread num
	// maybe should use a for loop in the case a thread has to do more than one thread. Will this ever occur?
	if (t_idx < prev_board_num) {
		int prev_board_start = t_idx * BOARD_SIZE;
		int* thread_prev_board = (int*)malloc(sizeof(int) * BOARD_SIZE);

		for (int i = 0; i < SUB_BOARD_SIZE; i++) { // read prev board into a sudoku sized local array
			thread_prev_board[i] = prev_boards[prev_board_start + i];
		}

		// find next index we can add to
		int empty_cell = FindNextEmptyCell(thread_prev_board);
		if (empty_cell == -1) { // Board is full
			return;
		}

		// Now try all possible numbers in this cell that islegal
		for (int i = 1; i <= 9; i++) {
			if (IsLegal(thread_prev_board, empty_cell, i)) { // number can go in this cell

				// where to start appending for the new board
				int new_board_offset = atomicAdd(&new_board_num, 1); // increment amount of boards at the new depth

				for (int j = 0; j < BOARD_SIZE; j++) {
					int ind = (new_board_offset * BOARD_SIZE) + j;

					new_boards[ind] = thread_prev_board[j];
				}
			}
		}
	}

}

__global__ void SolveBoard(int **_all_boards, int *_solved_board) {

}

void solve(int *board, int depth) {


	int *new_boards;
	int *old_boards;
	int *solution;
	int *board_num;
	int host_solution[SUB_BOARD_SIZE*SUB_BOARD_SIZE];

	int DEPTH = 7;

	const int memSize = 81 * pow(9, DEPTH);

	// alloc device memory
	hipMalloc(&new_boards, memSize * sizeof(int));
	hipMalloc(&old_boards, memSize * sizeof(int));
	hipMalloc(&solution, SUB_BOARD_SIZE * SUB_BOARD_SIZE * sizeof(int));
	hipMalloc(&board_num, sizeof(int));

	hipMemset(board_num, 0, sizeof(int));
	hipMemset(new_boards, 0, memSize * sizeof(int));
	hipMemset(old_boards, 0, memSize * sizeof(int));
	hipMemset(solution, 0, SUB_BOARD_SIZE * SUB_BOARD_SIZE * sizeof(int));

	for (int i = 0; i < depth; i++) {

		/// Generate Board
		/// Save New Boards
		/// Fix indices
		/// reiterate

	}
}

int main()
{
	// Solve medium sized board and go to depth 7 of BFS
	solve(test_board_medium, 7);
}
