#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <set>
#include "board.h"
#include <chrono>
#include <string>

// Function to return integer array of board state
__device__ int* board_to_ints(Board * _board)
{
	if (_board->board_to_int != nullptr) {
		delete _board->board_to_int;
	}

	_board->board_to_int = new int[BOARD_SIZE];
	for (int i = 0; i < BOARD_SIZE; i++) {
		if (_board->board[i][0] == true) {
			for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
				if (_board->board[i][j] == true) {
					_board->board_to_int[i] = j;
				}
			}
		}

		else {
			_board->board_to_int[i] = 0;
		}
	}

	return _board->board_to_int;
}

__device__ bool row_check(const int* _board, int _board_root, int _row, int _entry, int loc) 
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i] == _entry) 
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check(const int* _board, int _board_root, int _col, int _entry, int loc) 
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry(const int* _board, int _board_root, int _row, int _col, int _entry, int loc) 
{
	return row_check(_board, _board_root, _row, _entry, loc) &&
		column_check(_board, _board_root, _col, _entry, loc) &&
		grid_check(_board, _board_root, _row, _col, _entry, loc);
}

__device__ bool is_legal(Board * _board)
{
	for (int i = 0; i < BOARD_SIZE; i++)
	{
		int* int_board = board_to_ints(_board);
		int row = i / SUB_BOARD_SIZE;
		int col = i % SUB_BOARD_SIZE;

		if (int_board[i] != 0 && !is_legal_entry(int_board, SUB_BOARD_SIZE, row, col, int_board[i], i)) 
		{
			//print_board();
			//print_cell(i);
			//throw;
			return false;
		}
	}
	return true;
}

__device__ void update_potentials(Board * _board, int _loc, int _val)
{
	if (_board->board[_loc][0] == false) // dont do anything if cell is not filled
		return;

	int row = _loc / SUB_BOARD_SIZE;
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		int row_ind = row * SUB_BOARD_SIZE + i;
		if (row_ind != _loc) {
			_board->board[row_ind][_val] = false;
		}
	}

	int col = _loc % SUB_BOARD_SIZE;
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		int col_ind = col + (SUB_BOARD_SIZE * i);
		if (col_ind != _loc) {
			_board->board[col_ind][_val] = false;
		}
	}

	int sub_grid_x = row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != _loc) {
				_board->board[ind][_val] = false;
			}
		}
	}
}

// sets a cell using 1d coordinates
__device__ void set_cell(Board * _board, int _loc, int _val) {
	is_legal(_board);
	_board->board[_loc][0] = true;
	for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
		if (_board->board[_loc][i] == true && i != _val) {
			_board->board[_loc][i] = false;
		}
	}
	update_potentials(_board, _loc, _val);
	is_legal(_board);
	--_board->empty_cells;
}

//sets a cell
__device__ void set_cell(Board * _board, int _row, int _col, int _val)
{
	int board_cell = _row + _col * SUB_BOARD_SIZE;
	is_legal(_board);
	_board->board[board_cell][0] = true;
	for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
		if (_board->board[board_cell][i] == true && i != _val) {
			_board->board[board_cell][i] = false;
		}
	}
	update_potentials(_board, board_cell, _val);
	is_legal(_board);
	--_board->empty_cells;
}

__device__ int* get_potentials(Board * _board, int _loc) {
	if (_board->board[_loc][0] == false) {
		_board->to_pass = new int[SUB_BOARD_SIZE];
		for (int i = 0; i < SUB_BOARD_SIZE; i++) {
			if (_board->board[_loc][i] == true) {
				_board->to_pass[i] = i;
			}

			else {
				_board->to_pass[i] = 0;
			}
		}
	}

	return _board->to_pass;
}

// Helper method to get potential values in an unfilled cell
__device__ int * get_potential_set(Board * _board, int _loc, int &count) 
{
	count = 0;
	//Get count
	if (_board->board[_loc][0] == false) {
		for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
			if (_board->board[_loc][i] == true) 
			{
				count++;
			}
		}
	}
	//Get values
	int *vals = new int[count];
	count = 0;
	if (_board->board[_loc][0] == false) {
		for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
			if (_board->board[_loc][i] == true)
			{
				vals[count] = i; //Made need to reorder small -> largest if problems seen //////////////////////////////////////////////////////////////////
				count++;
			}
		}
	}
	return vals;
}

__device__ void annotate_potential_entries(Board * _board)
{
	is_legal(_board);

	//print_board();
	// std::cout << empty_cells << std::endl;
	for (int row = 0; row < SUB_BOARD_SIZE; row++) {
		// set to hold non-filled values in the row
		int * row_vals;
		int row_valsCount = 0;

		// std::cout << row_vals.size() << std::endl;
		// remove values from set that correspond to filled cells in the row
		for (int i = row * SUB_BOARD_SIZE; i < SUB_BOARD_SIZE + (row * SUB_BOARD_SIZE); i++) 
		{
			if (_board->board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) 
				{
					if (_board->board[i][j] == true) 
					{
						row_valsCount++;
					}
				}
			}
		}
		row_vals = new int[row_valsCount];
		row_valsCount = 0;
		for (int i = row * SUB_BOARD_SIZE; i < SUB_BOARD_SIZE + (row * SUB_BOARD_SIZE); i++)
		{
			if (_board->board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++)
				{
					if (_board->board[i][j] == true)
					{

						row_vals[row_valsCount] = j;
						row_valsCount++;
					}
				}
			}
		}

		// std::cout << "Got to 76" << std::endl;
		// Fill cells with true where indeces correspond to values it cannot have 
		if (row_valsCount > 0)
		{
			for (int i = row * SUB_BOARD_SIZE; i < (row * SUB_BOARD_SIZE) + SUB_BOARD_SIZE; i++) {
				if (_board->board[i][0] == false) 
				{
					for (int rowValue = 0; rowValue < row_valsCount -1; ++rowValue) ///////////////////////////////////////row_valsCount -1?
					{
						_board->board[i][row_vals[rowValue]] = false;
						is_legal(_board);
					}

					// check for single potential value and set if true
					int count = 0;
					int val = 0;
					for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
						if (_board->board[i][j] == true) {
							val = j;
							++count;
						}
					}
					if (count == 1) {
						set_cell(_board, i, val);
						is_legal(_board);
					}
				}
			}
		}
	}

	// std::cout << "Got to 101" << std::endl;
	// scan col for filled in values and store in temp set
	for (int col = 0; col < SUB_BOARD_SIZE; col++) 
	{
		int * col_vals;
		int col_valsCount = 0;
		for (int i = col; i < BOARD_SIZE; i += SUB_BOARD_SIZE) {
			// std::cout << "Got to 106" << std::endl;
			if (_board->board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
					if (_board->board[i][j] == true) 
					{
						col_valsCount++;
						break;
					}
				}
			}
		}
		col_vals = new int[col_valsCount];
		col_valsCount = 0;
		for (int i = col; i < BOARD_SIZE; i += SUB_BOARD_SIZE) {
			// std::cout << "Got to 106" << std::endl;
			if (_board->board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
					if (_board->board[i][j] == true)
					{
						col_vals[col_valsCount] = j;
						col_valsCount++;
						break;
					}
				}
			}
		}
		// std::cout << col_vals.size() << std::endl;
		// std::cout << "Got to 118" << std::endl;
		// Fill cells with true where indeces correspond to values it cannot have 
		if (col_valsCount > 0) 
		{
			for (int i = col; i < BOARD_SIZE; i += SUB_BOARD_SIZE) {
				if (_board->board[i][0] == false) 
				{
					for (int colsValue = 0; colsValue < col_valsCount -1; ++colsValue) ///////////////////////////////////////////////colsvalCount -1 ?
					{
						if (_board->board[i][col_vals[colsValue]] == true) {
							_board->board[i][col_vals[colsValue]] = false;
							is_legal(_board);
						}
					}

					// check for single potential value and set if true
					int count = 0;
					int val = 0;
					for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
						if (_board->board[i][j] == true) {
							val = j;
							++count;
						}
					}
					if (count == 1) {
						set_cell(_board, i, val);
						is_legal(_board);
					}
				}
			}
		}
	}
	// std::cout << "Got to 144" << std::endl;
	// Reduce potentials for sub grid intersections
	for (int grid_x = 0; grid_x < SUB_BOARD_DIM; grid_x++) {
		for (int grid_y = 0; grid_y < SUB_BOARD_DIM; grid_y++) 
		{
			int* grid_vals;
			int grid_valsCount = 0;
			int grid_start = grid_x * 9 * 3 + grid_y * 3;
			for (int row = 0; row < SUB_BOARD_DIM; row++) {
				for (int loc = grid_start + row * SUB_BOARD_SIZE; loc < (grid_start + row * SUB_BOARD_SIZE) + SUB_BOARD_DIM; loc++) {
					if (_board->board[loc][0] == true) {
						for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
							if (_board->board[loc][j] == true) 
							{
								grid_valsCount++;
								break;
							}
						}
					}
				}
			}
			grid_vals = new int[grid_valsCount];
			grid_valsCount = 0;
			grid_start = grid_x * 9 * 3 + grid_y * 3;
			for (int row = 0; row < SUB_BOARD_DIM; row++) {
				for (int loc = grid_start + row * SUB_BOARD_SIZE; loc < (grid_start + row * SUB_BOARD_SIZE) + SUB_BOARD_DIM; loc++) {
					if (_board->board[loc][0] == true) {
						for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
							if (_board->board[loc][j] == true)
							{
								grid_vals[grid_valsCount] = j;
								grid_valsCount++;
								break;
							}
						}
					}
				}
			}

			// std::cout << grid_vals.size() << std::endl;
			for (int row = 0; row < SUB_BOARD_DIM; row++) {
				for (int loc = grid_start + row * SUB_BOARD_SIZE; loc < (grid_start + row * SUB_BOARD_SIZE) + SUB_BOARD_DIM; loc++) {
					if (_board->board[loc][0] == false) 
					{
						for (int gridVal = 0; gridVal < grid_valsCount -1; ++gridVal) /////////////////////////////////////////////////////////////// grid_valsCount -1?
						{
							if (_board->board[loc][grid_vals[gridVal]] == true) {
								_board->board[loc][grid_vals[gridVal]] = false;
								is_legal(_board);
							}
						}

						// check for single potential value and set if true
						int count = 0;
						int val = 0;
						for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
							if (_board->board[loc][i] == true) {
								val = i;
								++count;
							}
						}
						if (count == 1) {
							_board->board[loc][0] = true;
							set_cell(_board, loc, val);
							is_legal(_board);
						}
					}
				}
			}
		}
	}
	// std::cout << empty_cells << std::endl;
}

__device__ void remove_potential_values(Board * _board, int* _vals, int _loc, int valCount) 
{
	if (_board->board[_loc][0] == false) 
	{
		for (int i = 0; i < valCount - 1; ++i) ////////////////////////////////////////////////valCount - 1? ////////////////////////////////////
		{
			_board->board[_loc][_vals[i]] = false;
		}
	}
}

// Helper to remove potential values from a row of a sub-grid assumes row_start is the leftmost cell of the row
__device__ void remove_potential_values_from_row(Board* _board, int* _vals, int row_start, int valCount) 
{
	remove_potential_values(_board, _vals, row_start, valCount);
	remove_potential_values(_board, _vals, row_start + 1, valCount);
	remove_potential_values(_board, _vals, row_start + 2, valCount);
}

// Helper to remove potential values from a col of a sub-grid assumes col_start is the topmost cell of the col
__device__ void remove_potential_values_from_col(Board* _board, int* _vals, int col_start, int valCount) {
	remove_potential_values(_board, _vals, col_start, valCount);
	remove_potential_values(_board, _vals, col_start + SUB_BOARD_SIZE, valCount);
	remove_potential_values(_board, _vals, col_start + SUB_BOARD_SIZE * 2, valCount);
}

// Helper method to remove specified potential values from a cell

__device__ void remove_doubles_and_triples_by_sub_grid(Board * _board)
{
	is_legal(_board);
	// Iterate by sub grid 
	for (int sub_grid_row = 0; sub_grid_row < SUB_BOARD_DIM; sub_grid_row++) {
		for (int sub_grid_col = 0; sub_grid_col < SUB_BOARD_DIM; sub_grid_col++) {

			// Iterate through sub-grid rows first
			int grid_start = (SUB_BOARD_DIM * SUB_BOARD_SIZE * sub_grid_row) + (SUB_BOARD_DIM * sub_grid_col);

			// For 9x9 sudoku
			if (SUB_BOARD_DIM == 3) {

				// For each row, we get the 3 potential sets for the cells
				for (int row = 0; row < SUB_BOARD_DIM; row++) 
				{
					int cell1Count = 0;
					int *cell_1 = get_potential_set(_board, row * SUB_BOARD_SIZE + grid_start, cell1Count);
					int cell2Count = 0;
					int *cell_2 = get_potential_set(_board, row * SUB_BOARD_SIZE + grid_start + 1, cell2Count);
					int cell3Count = 0;
					int *cell_3 = get_potential_set(_board, row * SUB_BOARD_SIZE + grid_start + 2, cell2Count);

					// check for triples
					if (cell1Count == 3 && cell2Count == 3 && cell3Count == 3)
					{
						//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////Potential problem area. Had to change cell_x to int*
						bool tripleFound = true;
						if (cell1Count == cell2Count && cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count] || cell_1[count] != cell_3[count])
								{
									tripleFound = false;
								}
							}
						}
						else
						{
							tripleFound = false;
						}

						if (tripleFound)
						{ // found a row triple

							// remove triple from all grid cells besides these 3
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) 
								{
									remove_potential_values_from_row(_board, cell_1, SUB_BOARD_SIZE * row_remove + grid_start, cell1Count);
								}
							}

							continue; // If triple was found, we don't want to waste time checking for a double
						}
					}

					// check for doubles - things gonna get a bit messy
					// start with cell 1
					if (cell1Count == 2)
					{
						bool doubleFoundC1C2 = true;
						if (cell1Count == cell2Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count])
								{
									doubleFoundC1C2 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C2 = false;
						}

						bool doubleFoundC1C3 = true;
						if (cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_3[count])
								{
									doubleFoundC1C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C3 = false;
						}

						if (doubleFoundC1C2)
						{ // double found
							// remove cell_3 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, row * SUB_BOARD_SIZE + grid_start + 2, cell2Count);
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) {
									remove_potential_values_from_row(_board, cell_2, SUB_BOARD_SIZE * row_remove + grid_start, cell2Count);
								}
							}
						}
						else if (doubleFoundC1C3) 
						{ // double found
							// remove cell_2 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, row * SUB_BOARD_SIZE + grid_start + 1, cell2Count);
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) {
									remove_potential_values_from_row(_board, cell_2, SUB_BOARD_SIZE * row_remove + grid_start, cell2Count);
								}
							}
						}
					} // cell 1 out of the running
					else if (cell2Count == 2 && cell3Count == 2)
					{
						bool doubleFoundC2C3 = true;
						if (cell2Count == cell3Count)
						{
							for (int count = 0; count < cell2Count; count++)
							{
								if (cell_2[count] != cell_3[count])
								{
									doubleFoundC2C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC2C3 = false;
						}

						if (doubleFoundC2C3)
						{ // double found
							// remove cell_1 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, row * SUB_BOARD_SIZE + grid_start, cell2Count);
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) {
									remove_potential_values_from_row(_board, cell_2, SUB_BOARD_SIZE * row_remove + grid_start, cell2Count);
								}
							}
						}
					}

					// no doubles or triples found on this row
				}

				// Now we do columns
				for (int col = 0; col < SUB_BOARD_DIM; col++) 
				{
					int cell1Count = 0;
					int*cell_1 = get_potential_set(_board, col + grid_start, cell1Count);
					int cell2Count = 0;
					int*cell_2 = get_potential_set(_board, col + grid_start + SUB_BOARD_SIZE, cell2Count);
					int cell3Count = 0;
					int*cell_3 = get_potential_set(_board, col + grid_start + SUB_BOARD_SIZE * 2, cell3Count);

					// check for triples
					if (cell1Count == 3 && cell2Count == 3 && cell3Count == 3) 
					{

						bool tripleFound = true;
						if (cell1Count == cell2Count && cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count] || cell_1[count] != cell_3[count])
								{
									tripleFound = false;
								}
							}
						}
						else
						{
							tripleFound = false;
						}


						if (tripleFound) { // found a row triple

							// remove triple from all grid cells besides these 3
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) 
							{
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_1, col_remove + grid_start, cell1Count);
								}
							}

							continue; // If triple was found, we don't want to waste time checking for a double
						}
					}

					// check for doubles - things gonna get a bit messy
					// start with cell 1
					if (cell1Count == 2) 
					{

						bool doubleFoundC1C2 = true;
						if (cell1Count == cell2Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count])
								{
									doubleFoundC1C2 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C2 = false;
						}

						bool doubleFoundC1C3 = true;
						if (cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_3[count])
								{
									doubleFoundC1C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C3 = false;
						}


						if (doubleFoundC1C2) 
						{ // double found
							// remove cell_3 vals and then the potential vals from other 2 cols
							remove_potential_values(_board, cell_2, col + grid_start + SUB_BOARD_SIZE * 2, cell2Count);
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) {
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_2, col_remove + grid_start, cell2Count);
								}
							}
						}
						else if (doubleFoundC1C3) 
						{ // double found
							// remove cell_2 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, col + grid_start + SUB_BOARD_SIZE, cell2Count);
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) {
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_2, col_remove + grid_start, cell2Count);
								}
							}
						}
					} // cell 1 out of the running
					else if (cell2Count == 2 && cell3Count == 2) 
					{
						bool doubleFoundC2C3 = true;
						if (cell2Count == cell3Count)
						{
							for (int count = 0; count < cell2Count; count++)
							{
								if (cell_2[count] != cell_3[count])
								{
									doubleFoundC2C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC2C3 = false;
						}

						if (doubleFoundC2C3) 
						{ // double found
							// remove cell_1 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, col + grid_start, cell2Count);
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) {
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_2, col_remove + grid_start, cell2Count);
								}
							}
						}
					}

					// no doubles or triples found on this col
				}
			}
			else {
				// TODO: Any other sudoku dimensions.
				//  e.g. 16x16 sudoku which will need to check for quadruples as well.
			}
		}
	}
}

__device__ void find_unique_cell_potential(Board * _board, int _loc)
{
	is_legal(_board);
	// do nothing if the board cell is already filled
	if (_board->board[_loc][0] == true)
		return;

	int pooled_potentialsCount = 0;
	int * pooled_potentials;
	int selected_potentialsCount = 0;
	int * selected_potentials = get_potential_set(_board, _loc, selected_potentialsCount);

	// Do rows first
	int row = _loc / SUB_BOARD_SIZE;
	// pool all row cell potentials besides the selected cell
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		int row_ind = row * SUB_BOARD_SIZE + i;
		if (row_ind != _loc) 
		{
			int cell_setCount = 0;
			int* cell_set = get_potential_set(_board, row_ind, cell_setCount);

			pooled_potentialsCount = cell_setCount;
			pooled_potentials = new int[pooled_potentialsCount];
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				pooled_potentials[j] = cell_set[j];
			}			
		}
	}

	// If not, perform set difference of first set w.r.t. pooled set
	if (pooled_potentialsCount > 0) 
	{
		//Need to find first number in selected not found
		int diff = 0;
		int diffCount = 0;
		for (int i = 0; i < selected_potentialsCount; i++)
		{
			bool found = false;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				if (selected_potentials[i] == pooled_potentials[j])
				{
					found = true;
					break;
				}
			}
			if (!found)
			{
				diff = selected_potentials[i];
				diffCount++;
			}
		}

		// only matters if we found a unique potential
		if (diffCount == 1)
		{
			set_cell(_board, _loc, diff);
			is_legal(_board);
			// cell is set now so we're done
			return;
		}
	}

	// Do cols next
	delete[] pooled_potentials;
	pooled_potentialsCount = 0;
	int col = _loc % SUB_BOARD_SIZE;
	// pool all col cell potentials besides the selected cell
	for (int i = 0; i < SUB_BOARD_SIZE; i++) 
	{
		int col_ind = col + (SUB_BOARD_SIZE * i);
		if (col_ind != _loc)
		{
			int cell_setCount = 0;
			int* cell_set = get_potential_set(_board, col_ind, cell_setCount);
			pooled_potentialsCount = cell_setCount;
			pooled_potentials = new int[pooled_potentialsCount];
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				pooled_potentials[j] = cell_set[j];
			}
		}
	}

	// If not, perform set difference of first set w.r.t. pooled set
	if (pooled_potentialsCount > 0) 
	{
		//Need to find first number in selected not found
		int diff = 0;
		int diffCount = 0;
		for (int i = 0; i < selected_potentialsCount; i++)
		{
			bool found = false;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				if (selected_potentials[i] == pooled_potentials[j])
				{
					found = true;
					break;
				}
			}
			if (!found)
			{
				diff = selected_potentials[i];
				diffCount++;
			}
		}

		// only matters if we found a unique potential
		if (diffCount == 1)
		{
			set_cell(_board, _loc, diff);
			is_legal(_board);
			// cell is set now so we're done
			return;
		}

	}

	// Finally, do sub grids 
	delete[] pooled_potentials;
	pooled_potentialsCount = 0;
	int sub_grid_x = row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != _loc) 
			{
				int cell_setCount = 0;
				int* cell_set = get_potential_set(_board, ind, cell_setCount);
				pooled_potentialsCount = cell_setCount;
				pooled_potentials = new int[pooled_potentialsCount];
				for (int j = 0; j < pooled_potentialsCount; j++)
				{
					pooled_potentials[j] = cell_set[j];
				}
			}
		}
	}
	// If not, perform set difference of first set w.r.t. pooled set
	if (pooled_potentialsCount > 0)
	{
		//Need to find first number in selected not found
		int diff = 0;
		int diffCount = 0;
		for (int i = 0; i < selected_potentialsCount; i++)
		{
			bool found = false;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				if (selected_potentials[i] == pooled_potentials[j])
				{
					found = true;
					break;
				}
			}
			if (!found)
			{
				diff = selected_potentials[i];
				diffCount++;
			}
		}

		// only matters if we found a unique potential
		if (diffCount == 1)
		{
			set_cell(_board, _loc, diff);
			is_legal(_board);
			// cell is set now so we're done
			return;
		}

	}
}

// performs unique potential on the entire board
__device__ void find_unique_potentials(Board * _board) 
{
	for (int i = 0; i < BOARD_SIZE; i++) 
	{
		find_unique_cell_potential(_board, i);
	}
}

__global__ void SudukoSolver(Board * _board)
{
	annotate_potential_entries(_board);
	remove_doubles_and_triples_by_sub_grid(_board);
	find_unique_potentials(_board);
}


#pragma region Boards
/*Boards*/
// https://www.puzzles.ca/sudoku_puzzles/sudoku_easy_487.html
int* test_board_easy = new int[81]{ 0, 7, 0, 0, 0, 1, 0, 0, 0,
									6, 0, 0, 0, 0, 0, 0, 0, 0,
									0, 0, 0, 0, 5, 3, 0, 0, 0,
									0, 0, 0, 8, 0, 0, 0, 2, 0,
									0, 3, 0, 0, 4, 7, 1, 6, 0,
									4, 0, 0, 0, 0, 0, 5, 7, 0,
									0, 0, 0, 0, 1, 0, 7, 5, 0,
									0, 6, 0, 5, 2, 0, 0, 4, 0,
									3, 0, 0, 0, 0, 9, 0, 8, 6 };

int* easy_test_answer = new int[81]{ 5, 7, 9, 6, 8, 1, 4, 3, 2,
									6, 2, 3, 7, 9, 4, 8, 1, 5,
									1, 8, 4, 2, 5, 3, 6, 9, 7,
									7, 1, 6, 8, 3, 5, 9, 2, 4,
									2, 3, 5, 9, 4, 7, 1, 6, 8,
									4, 9, 8, 1, 6, 2, 5, 7, 3,
									8, 4, 2, 3, 1, 6, 7, 5, 9,
									9, 6, 7, 5, 2, 8, 3, 4, 1,
									3, 5, 1, 4, 7, 9, 2, 8, 6 };

int* test_board_easy2 = new int[81]{ 0, 0, 0, 0, 9, 0, 0, 0, 0,
									  0, 3, 0, 0, 0, 0, 7, 4, 2,
									  6, 1, 0, 0, 3, 0, 0, 0, 0,
									  0, 0, 8, 0, 0, 0, 0, 0, 0,
									  0, 2, 9, 1, 4, 7, 0, 0, 8,
									  4, 5, 0, 0, 0, 0, 0, 7, 0,
									  0, 4, 0, 0, 0, 0, 0, 6, 0,
									  5, 0, 0, 6, 0, 0, 0, 3, 0,
									  0, 0, 0, 0, 8, 1, 2, 0, 0 };

int* easy_test2_answer = new int[81]{ 2, 8, 4, 7, 9, 5, 3, 1, 6,
									   9, 3, 5, 8, 1, 6, 7, 4, 2,
									   6, 1, 7, 2, 3, 4, 5, 8, 9,
									   1, 7, 8, 5, 6, 9, 4, 2, 3,
									   3, 2, 9, 1, 4, 7, 6, 5, 8,
									   4, 5, 6, 3, 2, 8, 9, 7, 1,
									   8, 4, 2, 9, 5, 3, 1, 6, 7,
									   5, 9, 1, 6, 7, 2, 8, 3, 4,
									   7, 6, 3, 4, 8, 1, 2, 9, 5 };

// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_medium = new int[81]{ 0, 0, 0, 0, 0, 0, 3, 0, 0,
									   8, 5, 2, 3, 0, 0, 0, 0, 1,
									   0, 9, 0, 2, 0, 0, 0, 0, 4,
									   9, 7, 4, 0, 0, 0, 0, 0, 0,
									   0, 1, 0, 0, 6, 0, 0, 0, 0,
									   0, 0, 0, 0, 4, 0, 0, 0, 0,
									   6, 0, 9, 0, 8, 0, 0, 3, 7,
									   3, 0, 0, 0, 0, 0, 0, 6, 0,
									   0, 2, 0, 0, 0, 5, 0, 0, 0 };

int* medium_test_answer = new int[81]{ 4, 6, 7, 9, 1, 8, 3, 2, 5,
										8, 5, 2, 3, 7, 4, 6 ,9, 1,
										1, 9, 3, 2, 5, 6, 7, 8, 4,
										9, 7, 4, 5, 2, 3, 8, 1, 6,
										2, 1, 8, 7, 6, 9, 4, 5, 3,
										5, 3, 6, 8, 4, 1, 2, 7, 9,
										6, 4, 9, 1, 8, 2, 5, 3, 7,
										3, 8, 5, 4, 9, 7, 1, 6, 2,
										7, 2, 1, 6, 3, 5, 9, 4, 8 };


// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_hard = new int[81]{ 0, 7, 0, 5, 0, 6, 0, 0, 0,
									 4, 0, 3, 0, 0, 0, 0, 0, 1,
									 0, 6, 0, 0, 0, 0, 9, 0, 7,
									 0, 0, 0, 7, 3, 0, 8, 2, 0,
									 8, 0, 4, 0, 5, 0, 0, 7, 3,
									 0, 9, 0, 0, 2, 0, 0, 0, 5,
									 0, 0, 1, 0, 0, 0, 0, 0, 0,
									 0, 0, 0, 1, 0, 0, 2, 0, 6,
									 0, 0, 0, 3, 8, 2, 0, 0, 0 };

//int* hard_test_answer = new int[81]{    4, 6, 7, 9, 1, 8, 3, 2, 5,
//										8, 5, 2, 3, 7, 4, 6 ,9, 1,
//										1, 9, 3, 2, 5, 6, 7, 8, 4,
//										9, 7, 4, 5, 2, 3, 8, 1, 6,
//										2, 1, 8, 7, 6, 9, 4, 5, 3,
//										5, 3, 6, 8, 4, 1, 2, 7, 9,
//										6, 4, 9, 1, 8, 2, 5, 3, 7,
//										3, 8, 5, 4, 9, 7, 1, 6, 2,
//										7, 2, 1, 6, 3, 5, 9, 4, 8 };

// http://www.ams.org/notices/200904/rtx090400460p.pdf
int* test_board_diabolical = new int[81]{ 0, 9, 0, 7, 0, 0, 8, 6, 0,
										  0, 3, 1, 0, 0, 5, 0, 2, 0,
										  8, 0, 6, 0, 0, 0, 0, 0, 0,
										  0, 0, 7, 0, 5, 0, 0, 0, 6,
										  0, 0, 0, 3, 0, 7, 0, 0, 0,
										  5, 0, 0, 0, 1, 0, 7, 0, 0,
										  0, 0, 0, 0, 0, 0, 1, 0, 9,
										  0, 2, 0, 6, 0, 0, 3, 5, 0,
										  0, 5, 4, 0, 0, 8, 0, 7, 0 };

int* diabolical_test_answer = new int[81]{ 2, 9, 5, 7, 4, 3, 8, 6, 1,
										   4, 3, 1, 8, 6, 5, 9, 2, 7,
										   8, 7, 6, 1, 9, 2, 5, 4, 3,
										   3, 8, 7, 4, 5, 9, 2, 1, 6,
										   6, 1, 2, 3, 8, 7, 4, 9, 5,
										   5, 4, 9, 2, 1, 6, 7, 3, 8,
										   7, 6, 3, 5, 2, 4, 1, 8, 9,
										   9, 2, 8, 6, 7, 1, 3, 5, 4,
										   1, 5, 4, 9, 3, 8, 6, 7, 2 };

#pragma endregion


Board *SetBoard()
{
	Board *board = new Board();
	board->set_board(test_board_easy);
	board->print_board();
	return board;
}

//Print timing of gpu memory and op timing as well as just op timing
void PrintTiming(float _opTime, float _memAndOpTime)
{
	std::cout << "\tMemory and Operation time: " << _memAndOpTime << " milliseconds." << std::endl;
	std::cout << "\tOperation time: " << _opTime << " milliseconds.\n" << std::endl;
}

void SolvePuzzle(Board *_board)
{
	hipEvent_t startMem, stopMem, startOp, stopOp;
	hipEventCreate(&startMem);
	hipEventCreate(&stopMem);
	hipEventCreate(&startOp);
	hipEventCreate(&stopOp);

	Board *device_board;
	int memorySize = BOARD_SIZE * sizeof(Board);

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	cudaStatus = hipMalloc((void **)&device_board, memorySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
	}



	int loop_count_easy = 0;
	//while (_board->is_complete() == false)
	//{

		// start memory + solver timing
		hipEventRecord(startMem);

		cudaStatus = hipMemcpy(device_board, _board, memorySize, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}

		//Start timing math only
		hipEventRecord(startOp);

		//Call Kernel
		SudukoSolver << <1, 1 >> > (device_board);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}

		//Stop puzzle only timing
		hipEventRecord(stopOp);

		// Copy result back to host
		hipMemcpy(_board, device_board, memorySize, hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!\n\n");
		}


		//Stop memory timing: sync must go here or it loses these timing events
		hipEventRecord(stopMem);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) 
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SudukoSolver!\n\n", cudaStatus);
		}


		float millisecondsOp = 0;
		float millisecondsMem = 0;
		hipEventElapsedTime(&millisecondsOp, startOp, stopOp);
		hipEventElapsedTime(&millisecondsMem, startMem, stopMem);

		//Print Timings
		PrintTiming(millisecondsOp, millisecondsMem);

	//	if (loop_count_easy++ > 15) {
	//		break;
	//	}

	//}
}

int main()
{
	// Instantiates, Sets, and Prints out the initial game board
	Board *easy_sudoku = SetBoard();
	int loop_count_easy = 0;

	SolvePuzzle(easy_sudoku);

	easy_sudoku->print_board();

	//std::cout << "Loops: " << loop_count_easy << " | Empty Cells: ";
	//std::cout << easy_sudoku->empty_cells << std::endl;



	/*while (easy_sudoku->is_complete() == false) 
	{
		easy_sudoku->annotate_potential_entries();
		easy_sudoku->remove_doubles_and_triples_by_sub_grid();
		easy_sudoku->find_unique_potentials();
		std::cout << "Loops: " << ++loop_count_easy << " | Empty Cells: ";
		std::cout << easy_sudoku->empty_cells << std::endl;
		if (loop_count_easy > 15) {
			break;
		}
	}


	std::cout << "Easy Board is correct: " << easy_sudoku->is_legal() << std::endl;*/
	//easy_sudoku->print_board();


}
