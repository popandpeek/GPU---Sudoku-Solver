#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <set>
#include "board.h"
#include <chrono>
#include <string>

// Function to return integer array of board state
//__device__ void board_to_ints(bool ** _board) //was int *
//{
	//if (_board->board_to_int != nullptr) 
	//{
	//	delete _board->board_to_int;
	//}
	//int * board_to_int = new int[BOARD_SIZE];
	//for (int i = 0; i < BOARD_SIZE; i++) 
	//{
	//	if (_board[i][0] == true) 
	//	{
	//		for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) 
	//		{
	//			if (_board[i][j] == true) 
	//			{
	//				board_to_int[i] = j;
	//			}
	//		}
	//	}
	//	else {
	//		board_to_int[i] = 0;
	//	}
	//}
	//return null;
//}

__device__ bool row_check_dev(const int* _board, int _board_root, int _row, int _entry, int loc) 
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i] == _entry) 
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check_dev(const int* _board, int _board_root, int _col, int _entry, int loc) 
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check_dev(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry_dev(const int* _board, int _board_root, int _row, int _col, int _entry, int loc) 
{
	return row_check_dev(_board, _board_root, _row, _entry, loc) &&
		column_check_dev(_board, _board_root, _col, _entry, loc) &&
		grid_check_dev(_board, _board_root, _row, _col, _entry, loc);
}

__device__ bool is_legal(bool ** _board)
{
	//for (int i = 0; i < BOARD_SIZE; i++)
	//{
	//	int* int_board = board_to_ints(_board);
	//	int row = i / SUB_BOARD_SIZE;
	//	int col = i % SUB_BOARD_SIZE;

	//	if (int_board[i] != 0 && !is_legal_entry(int_board, SUB_BOARD_SIZE, row, col, int_board[i], i)) 
	//	{
	//		//print_board();
	//		//print_cell(i);
	//		//throw;
	//		return false;
	//	}
	//}
	return true;
}

__device__ bool is_legal_1D(int *_board)
{
	for (int i = 0; i < BOARD_SIZE; i++) {
		int row = i / SUB_BOARD_SIZE;
		int col = i % SUB_BOARD_SIZE;

		if (_board[i] != 0 && !is_legal_entry_dev(_board, SUB_BOARD_SIZE, row, col, _board[i], i))
		{
			//print_board();
			//print_cell(i);
			//throw;
			return false;
		}
	}
	return true;
}

__device__ void update_potentials(bool ** _board, int _loc, int _val)
{
	if (_board[_loc][0] == false) // dont do anything if cell is not filled
		return;

	int row = _loc / SUB_BOARD_SIZE;
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		int row_ind = row * SUB_BOARD_SIZE + i;
		if (row_ind != _loc) {
			_board[row_ind][_val] = false;
		}
	}

	int col = _loc % SUB_BOARD_SIZE;
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		int col_ind = col + (SUB_BOARD_SIZE * i);
		if (col_ind != _loc) {
			_board[col_ind][_val] = false;
		}
	}

	int sub_grid_x = row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != _loc) {
				_board[ind][_val] = false;
			}
		}
	}
}

// sets a cell using 1d coordinates
__device__ void set_cell(bool ** _board, int _loc, int _val, int * _emptyCells) 
{
	is_legal(_board);
	_board[_loc][0] = true;
	for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
		if (_board[_loc][i] == true && i != _val) {
			_board[_loc][i] = false;
		}
	}
	update_potentials(_board, _loc, _val);
	is_legal(_board);
	--_emptyCells[0];
}

//sets a cell
__device__ void set_cell(bool ** _board, int _row, int _col, int _val, int * _emptyCells)
{
	int board_cell = _row + _col * SUB_BOARD_SIZE;
	is_legal(_board);
	_board[board_cell][0] = true;
	for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
		if (_board[board_cell][i] == true && i != _val) {
			_board[board_cell][i] = false;
		}
	}
	update_potentials(_board, board_cell, _val);
	is_legal(_board);
	--_emptyCells[0];
}

//__device__ int* get_potentials(bool ** _board, int _loc) 
//{
//	if (_board[_loc][0] == false) 
//	{
//		int *to_pass = new int[SUB_BOARD_SIZE];
//		for (int i = 0; i < SUB_BOARD_SIZE; i++) {
//			if (_board[_loc][i] == true) {
//				to_pass[i] = i;
//			}
//
//			else 
//			{
//				to_pass[i] = 0;
//			}
//		}
//	}
//
//	return to_pass;
//}

// Helper method to get potential values in an unfilled cell
__device__ int * get_potential_set(bool ** _board, int _loc, int &count, int * _pots_set)
{
	count = 0;

	for (int i = 0; i < SUB_BOARD_SIZE; i++)
	{
		_pots_set[i] = 0;
	}

	//Get count
	if (_board[_loc][0] == false) {
		for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
			if (_board[_loc][i] == true) 
			{
				count++;
			}
		}
	}
	//Get values

	count = 0;
	if (_board[_loc][0] == false) {
		for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
			if (_board[_loc][i] == true)
			{
				_pots_set[count] = i; //Made need to reorder small -> largest if problems seen //////////////////////////////////////////////////////////////////
				count++;
			}
		}
	}

	return _pots_set;
}

__device__ void annotate_potential_entries(bool ** _board, int * _emptyCells, int * _row_vals, int * _col_vals, int * _grid_vals)
{
	is_legal(_board);

	//print_board();
	// std::cout << empty_cells << std::endl;
	for (int row = 0; row < SUB_BOARD_SIZE; row++) {
		// set to hold non-filled values in the row

		int row_valsCount = 0;

		// std::cout << row_vals.size() << std::endl;
		// remove values from set that correspond to filled cells in the row
		for (int i = row * SUB_BOARD_SIZE; i < SUB_BOARD_SIZE + (row * SUB_BOARD_SIZE); i++) 
		{
			if (_board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) 
				{
					if (_board[i][j] == true) 
					{
						row_valsCount++;
					}
				}
			}
		}
	
		row_valsCount = 0;
		for (int i = row * SUB_BOARD_SIZE; i < SUB_BOARD_SIZE + (row * SUB_BOARD_SIZE); i++)
		{
			if (_board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++)
				{
					if (_board[i][j] == true)
					{

						_row_vals[row_valsCount] = j;
						row_valsCount++;
					}
				}
			}
		}

		// std::cout << "Got to 76" << std::endl;
		// Fill cells with true where indeces correspond to values it cannot have 
		if (row_valsCount > 0)
		{
			for (int i = row * SUB_BOARD_SIZE; i < (row * SUB_BOARD_SIZE) + SUB_BOARD_SIZE; i++) {
				if (_board[i][0] == false) 
				{
					for (int rowValue = 0; rowValue < row_valsCount; ++rowValue) ///////////////////////////////////////row_valsCount -1?
					{
						_board[i][_row_vals[rowValue]] = false;
						is_legal(_board);
					}

					// check for single potential value and set if true
					int count = 0;
					int val = 0;
					for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
						if (_board[i][j] == true) {
							val = j;
							++count;
						}
					}
					if (count == 1) {
						set_cell(_board, i, val, _emptyCells);
						is_legal(_board);
					}
				}
			}
		}
	}

	// std::cout << "Got to 101" << std::endl;
	// scan col for filled in values and store in temp set
	for (int col = 0; col < SUB_BOARD_SIZE; col++) 
	{
		int col_valsCount = 0;
		for (int i = col; i < BOARD_SIZE; i += SUB_BOARD_SIZE) {
			// std::cout << "Got to 106" << std::endl;
			if (_board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
					if (_board[i][j] == true) 
					{
						col_valsCount++;
						break;
					}
				}
			}
		}
		col_valsCount = 0;
		for (int i = col; i < BOARD_SIZE; i += SUB_BOARD_SIZE) {
			// std::cout << "Got to 106" << std::endl;
			if (_board[i][0] == true) {
				for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
					if (_board[i][j] == true)
					{
						_col_vals[col_valsCount] = j;
						col_valsCount++;
						break;
					}
				}
			}
		}
		// std::cout << col_vals.size() << std::endl;
		// std::cout << "Got to 118" << std::endl;
		// Fill cells with true where indeces correspond to values it cannot have 
		if (col_valsCount > 0) 
		{
			for (int i = col; i < BOARD_SIZE; i += SUB_BOARD_SIZE) {
				if (_board[i][0] == false) 
				{
					for (int colsValue = 0; colsValue < col_valsCount; ++colsValue) ///////////////////////////////////////////////colsvalCount -1 ?
					{
						if (_board[i][_col_vals[colsValue]] == true) {
							_board[i][_col_vals[colsValue]] = false;
							is_legal(_board);
						}
					}

					// check for single potential value and set if true
					int count = 0;
					int val = 0;
					for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
						if (_board[i][j] == true) {
							val = j;
							++count;
						}
					}
					if (count == 1) {
						set_cell(_board, i, val, _emptyCells);
						is_legal(_board);
					}
				}
			}
		}
	}
	// std::cout << "Got to 144" << std::endl;
	// Reduce potentials for sub grid intersections
	for (int grid_x = 0; grid_x < SUB_BOARD_DIM; grid_x++) {
		for (int grid_y = 0; grid_y < SUB_BOARD_DIM; grid_y++) 
		{
			int grid_valsCount = 0;
			int grid_start = grid_x * 9 * 3 + grid_y * 3;
			for (int row = 0; row < SUB_BOARD_DIM; row++) {
				for (int loc = grid_start + row * SUB_BOARD_SIZE; loc < (grid_start + row * SUB_BOARD_SIZE) + SUB_BOARD_DIM; loc++) {
					if (_board[loc][0] == true) {
						for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
							if (_board[loc][j] == true) 
							{
								grid_valsCount++;
								break;
							}
						}
					}
				}
			}
			grid_valsCount = 0;
			grid_start = grid_x * 9 * 3 + grid_y * 3;
			for (int row = 0; row < SUB_BOARD_DIM; row++) {
				for (int loc = grid_start + row * SUB_BOARD_SIZE; loc < (grid_start + row * SUB_BOARD_SIZE) + SUB_BOARD_DIM; loc++) {
					if (_board[loc][0] == true) {
						for (int j = 1; j < SUB_BOARD_SIZE + 1; j++) {
							if (_board[loc][j] == true)
							{
								_grid_vals[grid_valsCount] = j;
								grid_valsCount++;
								break;
							}
						}
					}
				}
			}

			// std::cout << grid_vals.size() << std::endl;
			for (int row = 0; row < SUB_BOARD_DIM; row++) {
				for (int loc = grid_start + row * SUB_BOARD_SIZE; loc < (grid_start + row * SUB_BOARD_SIZE) + SUB_BOARD_DIM; loc++) {
					if (_board[loc][0] == false) 
					{
						for (int gridVal = 0; gridVal < grid_valsCount; ++gridVal) /////////////////////////////////////////////////////////////// grid_valsCount -1?
						{
							if (_board[loc][_grid_vals[gridVal]] == true) {
								_board[loc][_grid_vals[gridVal]] = false;
								is_legal(_board);
							}
						}

						// check for single potential value and set if true
						int count = 0;
						int val = 0;
						for (int i = 1; i < SUB_BOARD_SIZE + 1; i++) {
							if (_board[loc][i] == true) {
								val = i;
								++count;
							}
						}
						if (count == 1) 
						{
							_board[loc][0] = true;
							set_cell(_board, loc, val, _emptyCells);
							is_legal(_board);
						}
					}
				}
			}
		}
	}
	// std::cout << empty_cells << std::endl;
}

__device__ void remove_potential_values(bool ** _board, int* _vals, int _loc, int valCount)
{
	if (_board[_loc][0] == false) 
	{
		for (int i = 0; i < valCount; ++i) ////////////////////////////////////////////////valCount - 1? ////////////////////////////////////
		{
			_board[_loc][_vals[i]] = false;
		}
	}
}

// Helper to remove potential values from a row of a sub-grid assumes row_start is the leftmost cell of the row
__device__ void remove_potential_values_from_row(bool ** _board, int* _vals, int row_start, int valCount)
{
	remove_potential_values(_board, _vals, row_start, valCount);
	remove_potential_values(_board, _vals, row_start + 1, valCount);
	remove_potential_values(_board, _vals, row_start + 2, valCount);
}

// Helper to remove potential values from a col of a sub-grid assumes col_start is the topmost cell of the col
__device__ void remove_potential_values_from_col(bool ** _board, int* _vals, int col_start, int valCount) {
	remove_potential_values(_board, _vals, col_start, valCount);
	remove_potential_values(_board, _vals, col_start + SUB_BOARD_SIZE, valCount);
	remove_potential_values(_board, _vals, col_start + SUB_BOARD_SIZE * 2, valCount);
}

// Helper method to remove specified potential values from a cell

__device__ void remove_doubles_and_triples_by_sub_grid(bool ** _board, int * _pots_set1, int * _pots_set2, int * _pots_set3)
{
	is_legal(_board);
	// Iterate by sub grid 
	for (int sub_grid_row = 0; sub_grid_row < SUB_BOARD_DIM; sub_grid_row++) {
		for (int sub_grid_col = 0; sub_grid_col < SUB_BOARD_DIM; sub_grid_col++) {

			// Iterate through sub-grid rows first
			int grid_start = (SUB_BOARD_DIM * SUB_BOARD_SIZE * sub_grid_row) + (SUB_BOARD_DIM * sub_grid_col);

			// For 9x9 sudoku
			if (SUB_BOARD_DIM == 3) {

				// For each row, we get the 3 potential sets for the cells
				for (int row = 0; row < SUB_BOARD_DIM; row++) 
				{
					int cell1Count = 0;
					int *cell_1 = get_potential_set(_board, row * SUB_BOARD_SIZE + grid_start, cell1Count, _pots_set1);
					int cell2Count = 0;
					int *cell_2 = get_potential_set(_board, row * SUB_BOARD_SIZE + grid_start + 1, cell2Count, _pots_set2);
					int cell3Count = 0;
					int *cell_3 = get_potential_set(_board, row * SUB_BOARD_SIZE + grid_start + 2, cell3Count, _pots_set3);

					// check for triples
					if (cell1Count == 3 && cell2Count == 3 && cell3Count == 3)
					{
						//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////Potential problem area. Had to change cell_x to int*
						bool tripleFound = true;
						if (cell1Count == cell2Count && cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count] || cell_1[count] != cell_3[count])
								{
									tripleFound = false;
								}
							}
						}
						else
						{
							tripleFound = false;
						}

						if (tripleFound)
						{ // found a row triple

							// remove triple from all grid cells besides these 3
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) 
								{
									remove_potential_values_from_row(_board, cell_1, SUB_BOARD_SIZE * row_remove + grid_start, cell1Count);
								}
							}

							continue; // If triple was found, we don't want to waste time checking for a double
						}
					}

					// check for doubles - things gonna get a bit messy
					// start with cell 1
					if (cell1Count == 2)
					{
						bool doubleFoundC1C2 = true;
						if (cell1Count == cell2Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count])
								{
									doubleFoundC1C2 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C2 = false;
						}

						bool doubleFoundC1C3 = true;
						if (cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_3[count])
								{
									doubleFoundC1C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C3 = false;
						}

						if (doubleFoundC1C2)
						{ // double found
							// remove cell_3 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, row * SUB_BOARD_SIZE + grid_start + 2, cell2Count);
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) {
									remove_potential_values_from_row(_board, cell_2, SUB_BOARD_SIZE * row_remove + grid_start, cell2Count);
								}
							}
						}
						else if (doubleFoundC1C3) 
						{ // double found
							// remove cell_2 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, row * SUB_BOARD_SIZE + grid_start + 1, cell2Count);
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) {
									remove_potential_values_from_row(_board, cell_2, SUB_BOARD_SIZE * row_remove + grid_start, cell2Count);
								}
							}
						}
					} // cell 1 out of the running
					else if (cell2Count == 2 && cell3Count == 2)
					{
						bool doubleFoundC2C3 = true;
						if (cell2Count == cell3Count)
						{
							for (int count = 0; count < cell2Count; count++)
							{
								if (cell_2[count] != cell_3[count])
								{
									doubleFoundC2C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC2C3 = false;
						}

						if (doubleFoundC2C3)
						{ // double found
							// remove cell_1 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, row * SUB_BOARD_SIZE + grid_start, cell2Count);
							for (int row_remove = 0; row_remove < SUB_BOARD_DIM; row_remove++) {
								if (row_remove != row) {
									remove_potential_values_from_row(_board, cell_2, SUB_BOARD_SIZE * row_remove + grid_start, cell2Count);
								}
							}
						}
					}

					// no doubles or triples found on this row
				}

				// Now we do columns
				for (int col = 0; col < SUB_BOARD_DIM; col++) 
				{
					int cell1Count = 0;
					int*cell_1 = get_potential_set(_board, col + grid_start, cell1Count, _pots_set1);
					int cell2Count = 0;
					int*cell_2 = get_potential_set(_board, col + grid_start + SUB_BOARD_SIZE, cell2Count, _pots_set2);
					int cell3Count = 0;
					int*cell_3 = get_potential_set(_board, col + grid_start + SUB_BOARD_SIZE * 2, cell3Count, _pots_set3);

					// check for triples
					if (cell1Count == 3 && cell2Count == 3 && cell3Count == 3) 
					{

						bool tripleFound = true;
						if (cell1Count == cell2Count && cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count] || cell_1[count] != cell_3[count])
								{
									tripleFound = false;
								}
							}
						}
						else
						{
							tripleFound = false;
						}


						if (tripleFound) { // found a row triple

							// remove triple from all grid cells besides these 3
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) 
							{
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_1, col_remove + grid_start, cell1Count);
								}
							}

							continue; // If triple was found, we don't want to waste time checking for a double
						}
					}

					// check for doubles - things gonna get a bit messy
					// start with cell 1
					if (cell1Count == 2) 
					{

						bool doubleFoundC1C2 = true;
						if (cell1Count == cell2Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_2[count])
								{
									doubleFoundC1C2 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C2 = false;
						}

						bool doubleFoundC1C3 = true;
						if (cell1Count == cell3Count)
						{
							for (int count = 0; count < cell1Count; count++)
							{
								if (cell_1[count] != cell_3[count])
								{
									doubleFoundC1C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC1C3 = false;
						}


						if (doubleFoundC1C2) 
						{ // double found
							// remove cell_3 vals and then the potential vals from other 2 cols
							remove_potential_values(_board, cell_2, col + grid_start + SUB_BOARD_SIZE * 2, cell2Count);
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) {
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_2, col_remove + grid_start, cell2Count);
								}
							}
						}
						else if (doubleFoundC1C3) 
						{ // double found
							// remove cell_2 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, col + grid_start + SUB_BOARD_SIZE, cell2Count);
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) {
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_2, col_remove + grid_start, cell2Count);
								}
							}
						}
					} // cell 1 out of the running
					else if (cell2Count == 2 && cell3Count == 2) 
					{
						bool doubleFoundC2C3 = true;
						if (cell2Count == cell3Count)
						{
							for (int count = 0; count < cell2Count; count++)
							{
								if (cell_2[count] != cell_3[count])
								{
									doubleFoundC2C3 = false;
								}
							}
						}
						else
						{
							doubleFoundC2C3 = false;
						}

						if (doubleFoundC2C3) 
						{ // double found
							// remove cell_1 vals and then the potential vals from other 2 rows
							remove_potential_values(_board, cell_2, col + grid_start, cell2Count);
							for (int col_remove = 0; col_remove < SUB_BOARD_DIM; col_remove++) {
								if (col_remove != col) {
									remove_potential_values_from_col(_board, cell_2, col_remove + grid_start, cell2Count);
								}
							}
						}
					}

					// no doubles or triples found on this col
				}
			}
			else {
				// TODO: Any other sudoku dimensions.
				//  e.g. 16x16 sudoku which will need to check for quadruples as well.
			}
		}
	}
}

__device__ void find_unique_cell_potential(bool ** _board, int _loc, int *_emptyCells, int * _pots_set, int *pots_set2, int * _pooled_pots)
{
	is_legal(_board);
	// do nothing if the board cell is already filled
	if (_board[_loc][0] == true)
		return;

	int pooled_potentialsCount = 0;
	int selected_potentialsCount = 0;
	int * selected_potentials = get_potential_set(_board, _loc, selected_potentialsCount, _pots_set);

	// Do rows first
	int row = _loc / SUB_BOARD_SIZE;
	// pool all row cell potentials besides the selected cell
	for (int i = 0; i < SUB_BOARD_SIZE; i++) {
		int row_ind = row * SUB_BOARD_SIZE + i;
		if (row_ind != _loc) 
		{
			int cell_setCount = 0;
			int* cell_set = get_potential_set(_board, row_ind, cell_setCount, pots_set2);

			pooled_potentialsCount = cell_setCount;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				_pooled_pots[j] = cell_set[j];
			}			
		}
	}

	// If not, perform set difference of first set w.r.t. pooled set
	if (pooled_potentialsCount > 0) 
	{
		//Need to find first number in selected not found
		int diff = 0;
		int diffCount = 0;
		for (int i = 0; i < selected_potentialsCount; i++)
		{
			bool found = false;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				if (selected_potentials[i] == _pooled_pots[j])
				{
					found = true;
					break;
				}
			}
			if (!found)
			{
				diff = selected_potentials[i];
				diffCount++;
			}
		}

		// only matters if we found a unique potential
		if (diffCount == 1)
		{
			set_cell(_board, _loc, diff, _emptyCells);
			is_legal(_board);
			// cell is set now so we're done
			return;
		}
	}

	// Do cols next
	pooled_potentialsCount = 0;
	int col = _loc % SUB_BOARD_SIZE;
	// pool all col cell potentials besides the selected cell
	for (int i = 0; i < SUB_BOARD_SIZE; i++) 
	{
		int col_ind = col + (SUB_BOARD_SIZE * i);
		if (col_ind != _loc)
		{
			int cell_setCount = 0;
			int* cell_set = get_potential_set(_board, col_ind, cell_setCount, pots_set2);
			pooled_potentialsCount = cell_setCount;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				_pooled_pots[j] = cell_set[j];
			}
		}
	}

	// If not, perform set difference of first set w.r.t. pooled set
	if (pooled_potentialsCount > 0) 
	{
		//Need to find first number in selected not found
		int diff = 0;
		int diffCount = 0;
		for (int i = 0; i < selected_potentialsCount; i++)
		{
			bool found = false;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				if (selected_potentials[i] == _pooled_pots[j])
				{
					found = true;
					break;
				}
			}
			if (!found)
			{
				diff = selected_potentials[i];
				diffCount++;
			}
		}

		// only matters if we found a unique potential
		if (diffCount == 1)
		{
			set_cell(_board, _loc, diff, _emptyCells);
			is_legal(_board);
			// cell is set now so we're done
			return;
		}

	}

	// Finally, do sub grids 
	pooled_potentialsCount = 0;
	int sub_grid_x = row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != _loc) 
			{
				int cell_setCount = 0;
				int* cell_set = get_potential_set(_board, ind, cell_setCount, pots_set2);
				pooled_potentialsCount = cell_setCount;
				for (int j = 0; j < pooled_potentialsCount; j++)
				{
					_pooled_pots[j] = cell_set[j];
				}
			}
		}
	}
	// If not, perform set difference of first set w.r.t. pooled set
	if (pooled_potentialsCount > 0)
	{
		//Need to find first number in selected not found
		int diff = 0;
		int diffCount = 0;
		for (int i = 0; i < selected_potentialsCount; i++)
		{
			bool found = false;
			for (int j = 0; j < pooled_potentialsCount; j++)
			{
				if (selected_potentials[i] == _pooled_pots[j])
				{
					found = true;
					break;
				}
			}
			if (!found)
			{
				diff = selected_potentials[i];
				diffCount++;
			}
		}

		// only matters if we found a unique potential
		if (diffCount == 1)
		{
			set_cell(_board, _loc, diff, _emptyCells);
			is_legal(_board);
			// cell is set now so we're done
			return;
		}

	}
}

void print_board_1d(int *board) {

	char* border = new char[26]{ "|-------+-------+-------|" };

	std::cout << border << std::endl;
	int split = sqrt(9);
	for (int i = 0; i < 9 * 9; i++) {
		if (i % 9 == 0) {
			std::cout << "| ";
		}
		else if (i % split == 0) {
			std::cout << "| ";
		}

		int value = board[i];
		if (value != 0) {
			std::cout << value << " ";
		}
		else {
			std::cout << ". ";
		}

		if (i % 9 == 9 - 1) {
			std::cout << "|" << std::endl;

			if (((i + 1) % (9 * 9 / split)) == 0) {
				std::cout << border << std::endl;
			}
		}
	}
	std::cout << std::endl;
}

// performs unique potential on the entire board
__device__ void find_unique_potentials(bool ** _board, int * _emptyCells, int* _pots_set, int* pots_set2, int * _pooled_pots)
{
	for (int i = 0; i < BOARD_SIZE; i++) 
	{
		find_unique_cell_potential(_board, i, _emptyCells, _pots_set, pots_set2, _pooled_pots);
	}
}

__global__ void CrooksSolver(bool ** _board, int *_emptyCells, int * _row_vals, int * _col_vals, int * _grid_vals, int * _pooled_pots)
{
	annotate_potential_entries(_board, _emptyCells, _row_vals, _col_vals, _grid_vals);
	remove_doubles_and_triples_by_sub_grid(_board, _row_vals, _col_vals, _grid_vals);
	//find_unique_potentials(_board, _emptyCells, _row_vals, _col_vals, _pooled_pots); //If we can get this function to work we can get crooks to work
}

__host__ std::stack<int*> SpawnBoards(Board *_board)
{
	int* start_state_board = _board->board_to_ints();
	std::stack<int*> board_stack;
	board_stack.push(start_state_board);

	int count = 0;

	while (true) {
		// Get the current state of board we will work on
		int* curr_board = board_stack.top();
		int next_cell = _board->find_next_empty_cell(curr_board);
		if (next_cell == -1) {
			curr_board = nullptr;
			break;
		}

		board_stack.pop();

		std::set<int> potential_values = _board->get_potential_set(next_cell);

		for (auto it = potential_values.begin(); it != potential_values.end(); it++) {
			int* next_board = _board->create_copy(curr_board);
			next_board[next_cell] = *it;
			board_stack.push(next_board);
			++count;
		}
	}

	//// allocate memory for returning array of boards to run on GPU
	//int* ret_boards = (int *)malloc((BOARD_SIZE)* count * sizeof(int));

	//// pop board_stack into array
	//for (int i = 0; i < count; i++) {
	//	ret_boards[i] = *board_stack.top();
	//	board_stack.pop();
	//}

	return board_stack;
}

__global__ void BackTracker(int * all_boards, int * solved_board)
{
	//Grab a board from all_boards
	//Find an empty cell
	//Attempt a number
		//If valid, substract from empty cell count and find next empty cell
		//If not valid, try the other numbers
		//If no number is valid, increase empty cell count and return to the last empty cell tried
			//To return to last empty cell tried we need to have a list of all the positions of the empty cells and use this to find the next empty cell so we don't get lost
	//If no more empty cells, set the solved_board and inform other threads via a shared memory variable or some other method that a solution was found and for them to quit
	//If stuck, then quit
}

__global__ void ValidBoards(int *_all_boards, int *_solved_board) {
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (is_legal_1D(&_all_boards[t_idx])) {
		for (int i = 0; i < 81; i++) {
			_solved_board[i] = _all_boards[i];
		}
	}
}

#pragma region Boards
/*Boards*/
// https://www.puzzles.ca/sudoku_puzzles/sudoku_easy_487.html
int* test_board_easy = new int[81]{ 0, 7, 0, 0, 0, 1, 0, 0, 0,
									6, 0, 0, 0, 0, 0, 0, 0, 0,
									0, 0, 0, 0, 5, 3, 0, 0, 0,
									0, 0, 0, 8, 0, 0, 0, 2, 0,
									0, 3, 0, 0, 4, 7, 1, 6, 0,
									4, 0, 0, 0, 0, 0, 5, 7, 0,
									0, 0, 0, 0, 1, 0, 7, 5, 0,
									0, 6, 0, 5, 2, 0, 0, 4, 0,
									3, 0, 0, 0, 0, 9, 0, 8, 6 };

int* easy_test_answer = new int[81]{ 5, 7, 9, 6, 8, 1, 4, 3, 2,
									6, 2, 3, 7, 9, 4, 8, 1, 5,
									1, 8, 4, 2, 5, 3, 6, 9, 7,
									7, 1, 6, 8, 3, 5, 9, 2, 4,
									2, 3, 5, 9, 4, 7, 1, 6, 8,
									4, 9, 8, 1, 6, 2, 5, 7, 3,
									8, 4, 2, 3, 1, 6, 7, 5, 9,
									9, 6, 7, 5, 2, 8, 3, 4, 1,
									3, 5, 1, 4, 7, 9, 2, 8, 6 };

int* test_board_easy2 = new int[81]{ 0, 0, 0, 0, 9, 0, 0, 0, 0,
									  0, 3, 0, 0, 0, 0, 7, 4, 2,
									  6, 1, 0, 0, 3, 0, 0, 0, 0,
									  0, 0, 8, 0, 0, 0, 0, 0, 0,
									  0, 2, 9, 1, 4, 7, 0, 0, 8,
									  4, 5, 0, 0, 0, 0, 0, 7, 0,
									  0, 4, 0, 0, 0, 0, 0, 6, 0,
									  5, 0, 0, 6, 0, 0, 0, 3, 0,
									  0, 0, 0, 0, 8, 1, 2, 0, 0 };

int* easy_test2_answer = new int[81]{ 2, 8, 4, 7, 9, 5, 3, 1, 6,
									   9, 3, 5, 8, 1, 6, 7, 4, 2,
									   6, 1, 7, 2, 3, 4, 5, 8, 9,
									   1, 7, 8, 5, 6, 9, 4, 2, 3,
									   3, 2, 9, 1, 4, 7, 6, 5, 8,
									   4, 5, 6, 3, 2, 8, 9, 7, 1,
									   8, 4, 2, 9, 5, 3, 1, 6, 7,
									   5, 9, 1, 6, 7, 2, 8, 3, 4,
									   7, 6, 3, 4, 8, 1, 2, 9, 5 };

// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_medium = new int[81]{ 0, 0, 0, 0, 0, 0, 3, 0, 0,
									   8, 5, 2, 3, 0, 0, 0, 0, 1,
									   0, 9, 0, 2, 0, 0, 0, 0, 4,
									   9, 7, 4, 0, 0, 0, 0, 0, 0,
									   0, 1, 0, 0, 6, 0, 0, 0, 0,
									   0, 0, 0, 0, 4, 0, 0, 0, 0,
									   6, 0, 9, 0, 8, 0, 0, 3, 7,
									   3, 0, 0, 0, 0, 0, 0, 6, 0,
									   0, 2, 0, 0, 0, 5, 0, 0, 0 };

int* medium_test_answer = new int[81]{ 4, 6, 7, 9, 1, 8, 3, 2, 5,
										8, 5, 2, 3, 7, 4, 6 ,9, 1,
										1, 9, 3, 2, 5, 6, 7, 8, 4,
										9, 7, 4, 5, 2, 3, 8, 1, 6,
										2, 1, 8, 7, 6, 9, 4, 5, 3,
										5, 3, 6, 8, 4, 1, 2, 7, 9,
										6, 4, 9, 1, 8, 2, 5, 3, 7,
										3, 8, 5, 4, 9, 7, 1, 6, 2,
										7, 2, 1, 6, 3, 5, 9, 4, 8 };


// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_hard = new int[81]{ 0, 7, 0, 5, 0, 6, 0, 0, 0,
									 4, 0, 3, 0, 0, 0, 0, 0, 1,
									 0, 6, 0, 0, 0, 0, 9, 0, 7,
									 0, 0, 0, 7, 3, 0, 8, 2, 0,
									 8, 0, 4, 0, 5, 0, 0, 7, 3,
									 0, 9, 0, 0, 2, 0, 0, 0, 5,
									 0, 0, 1, 0, 0, 0, 0, 0, 0,
									 0, 0, 0, 1, 0, 0, 2, 0, 6,
									 0, 0, 0, 3, 8, 2, 0, 0, 0 };

//int* hard_test_answer = new int[81]{    4, 6, 7, 9, 1, 8, 3, 2, 5,
//										8, 5, 2, 3, 7, 4, 6 ,9, 1,
//										1, 9, 3, 2, 5, 6, 7, 8, 4,
//										9, 7, 4, 5, 2, 3, 8, 1, 6,
//										2, 1, 8, 7, 6, 9, 4, 5, 3,
//										5, 3, 6, 8, 4, 1, 2, 7, 9,
//										6, 4, 9, 1, 8, 2, 5, 3, 7,
//										3, 8, 5, 4, 9, 7, 1, 6, 2,
//										7, 2, 1, 6, 3, 5, 9, 4, 8 };

// http://www.ams.org/notices/200904/rtx090400460p.pdf
int* test_board_diabolical = new int[81]{ 0, 9, 0, 7, 0, 0, 8, 6, 0,
										  0, 3, 1, 0, 0, 5, 0, 2, 0,
										  8, 0, 6, 0, 0, 0, 0, 0, 0,
										  0, 0, 7, 0, 5, 0, 0, 0, 6,
										  0, 0, 0, 3, 0, 7, 0, 0, 0,
										  5, 0, 0, 0, 1, 0, 7, 0, 0,
										  0, 0, 0, 0, 0, 0, 1, 0, 9,
										  0, 2, 0, 6, 0, 0, 3, 5, 0,
										  0, 5, 4, 0, 0, 8, 0, 7, 0 };

int* diabolical_test_answer = new int[81]{ 2, 9, 5, 7, 4, 3, 8, 6, 1,
										   4, 3, 1, 8, 6, 5, 9, 2, 7,
										   8, 7, 6, 1, 9, 2, 5, 4, 3,
										   3, 8, 7, 4, 5, 9, 2, 1, 6,
										   6, 1, 2, 3, 8, 7, 4, 9, 5,
										   5, 4, 9, 2, 1, 6, 7, 3, 8,
										   7, 6, 3, 5, 2, 4, 1, 8, 9,
										   9, 2, 8, 6, 7, 1, 3, 5, 4,
										   1, 5, 4, 9, 3, 8, 6, 7, 2 };

#pragma endregion


Board *SetBoard(int * _board)
{
	Board *board = new Board();
	board->set_board(_board);
	board->print_board();
	return board;
}

//Print timing of gpu memory and op timing as well as just op timing
void PrintTiming(float _opTime, float _memAndOpTime)
{
	std::cout << "\tMemory and Operation time: " << _memAndOpTime << " milliseconds." << std::endl;
	std::cout << "\tOperation time: " << _opTime << " milliseconds.\n" << std::endl;
}

int ReducePossibilities(Board *_board)
{
	hipEvent_t startMem, stopMem, startOp, stopOp;
	hipEventCreate(&startMem);
	hipEventCreate(&stopMem);
	hipEventCreate(&startOp);
	hipEventCreate(&stopOp);

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	bool *hostTempBoard[BOARD_SIZE];
	bool **device_board;
	hipMalloc((void **)&device_board, (BOARD_SIZE) * sizeof(bool *));
	for (int i = 0; i < BOARD_SIZE; i++)
	{
		hipMalloc((void **)&hostTempBoard[i], (SUB_BOARD_SIZE + 1) * sizeof(bool));
	}

	int * emptyCells = (int *)malloc(sizeof(int));
	int previousEmptyCells;
	emptyCells[0] = _board->empty_cells;
	int * device_emptyCells;
	hipMalloc((void**)&device_emptyCells, sizeof(int));

	int * row_val;
	hipMalloc((void**)&row_val, SUB_BOARD_SIZE * sizeof(int));
	int * col_val;
	hipMalloc((void**)&col_val, SUB_BOARD_SIZE * sizeof(int));
	int * grid_val;
	hipMalloc((void**)&grid_val, SUB_BOARD_SIZE * sizeof(int));

	int * pooled_pots;
	hipMalloc((void**)&pooled_pots, SUB_BOARD_SIZE * sizeof(int));

	float totalOpTime = 0;
	float totalMemTime = 0;

	while (true)
	{
		previousEmptyCells = emptyCells[0];
		// start memory + solver timing
		hipEventRecord(startMem);

		//Copy 2D bool array to device
		cudaStatus = hipMemcpy(device_board, hostTempBoard, (BOARD_SIZE) * sizeof(bool *), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}
		for (int i = 0; i < BOARD_SIZE; i++)
		{
			cudaStatus = hipMemcpy(hostTempBoard[i], _board->board[i], (SUB_BOARD_SIZE + 1) * sizeof(bool), hipMemcpyHostToDevice);
			if (cudaStatus != hipSuccess) {
				fprintf(stderr, "hipMemcpy failed!");
			}
		}

		//Copy emptyCell count over
		cudaStatus = hipMemcpy(device_emptyCells, emptyCells, sizeof(int), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
		}


		//Start timing math only
		hipEventRecord(startOp);





		//Call Kernel
		CrooksSolver << <1, 1 >> > (device_board, device_emptyCells, row_val, col_val, grid_val, pooled_pots);





		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		}

		//Stop puzzle only timing
		hipEventRecord(stopOp);

		//Copy back to host
		for (int i = 0; i < BOARD_SIZE; i++)
		{
			hipMemcpy(_board->board[i], hostTempBoard[i], (SUB_BOARD_SIZE + 1) * sizeof(bool), hipMemcpyDeviceToHost);
		}

		cudaStatus = hipMemcpy(emptyCells, device_emptyCells, sizeof(int), hipMemcpyDeviceToHost);



		//Stop memory timing: sync must go here or it loses these timing events
		hipEventRecord(stopMem);
		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) 
		{
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SudukoSolver!\n\n", cudaStatus);
		}


		float millisecondsOp = 0;
		float millisecondsMem = 0;
		hipEventElapsedTime(&millisecondsOp, startOp, stopOp);
		hipEventElapsedTime(&millisecondsMem, startMem, stopMem);

		totalOpTime += millisecondsOp;
		totalMemTime += millisecondsMem;

		if (emptyCells[0] == 0)
		{
			//Print Timings
			std::cout << "Reduce Possibilities Time:\n";
			PrintTiming(totalOpTime, totalMemTime);
			return 0;
		}
		if (emptyCells[0] == previousEmptyCells)
		{
			//Print Timings
			std::cout << "Reduce Possibilities Time:\n";
			PrintTiming(totalOpTime, totalMemTime);
			return previousEmptyCells;
		}

	}
	return previousEmptyCells;
}

bool BackTrack(Board * _board, int emptyCells)
{
	//WARNING: current DS does not hold potential values - only actual values


	hipEvent_t startMem, stopMem, startOp, stopOp;
	hipEventCreate(&startMem);
	hipEventCreate(&stopMem);
	hipEventCreate(&startOp);
	hipEventCreate(&stopOp);

	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}

	std::stack<int*> spawned = SpawnBoards(_board);
	int size = spawned.size();
	int **all_boards = new int*[size];
	for (int i = 0; i < spawned.size(); i++) {
		all_boards[i] = new int[81];
	}
	//int *host_all_boards = (int *)malloc((BOARD_SIZE) * spawned.size() * sizeof(int));
	int *host_answer_board = (int *)malloc(BOARD_SIZE * sizeof(int));
	int *device_answer_board;

	// Transfer stack of boards to array

	for (int i = 0; i < size; i++) {
			
		for (int j = 0; j < 81; j++) {
			int* next_board = spawned.top();
			all_boards[i][j] = next_board[j];
			std::cout << all_boards[i][j] << std::endl;
		}
		spawned.pop();
	}

	for (int i = 0; i < size; i++) {
		hipMalloc((void **)&all_boards[i], (BOARD_SIZE) * sizeof(int));
	}
	//int * host_start_board = _board->board_to_ints(); //Convert _board into ints
	hipMalloc((void **)&all_boards, (BOARD_SIZE) * spawned.size() * sizeof(int));
	hipMalloc((void **)&device_answer_board, (BOARD_SIZE) * sizeof(int));

	// start memory + solver timing
	hipEventRecord(startMem);

	////Copy 2D bool array to device
	//cudaStatus = hipMemcpy(device_start_board, host_start_board, (BOARD_SIZE) * sizeof(int), hipMemcpyHostToDevice);
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "hipMemcpy failed!");
	//}

	//Start timing math only
	hipEventRecord(startOp);

	int threads = 32;
	int blocks = (spawned.size() * SUB_BOARD_SIZE / threads) + 1;
	dim3 dimGrid(1, 1);
	dim3 dimBlock(blocks, threads);

	////Call Kernel
	//ValidBoards<<<1, 1>>>(all_boards, device_answer_board);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
	}

	//Stop puzzle only timing
	hipEventRecord(stopOp);

	//Copy back to host
	hipMemcpy(host_answer_board, device_answer_board, (BOARD_SIZE) * spawned.size() * sizeof(int), hipMemcpyDeviceToHost);

	print_board_1d(host_answer_board);

	Board *ans = new Board();
	ans->set_board(host_answer_board);
	ans->print_board();

	//Stop memory timing: sync must go here or it loses these timing events
	hipEventRecord(stopMem);
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SudukoSolver!\n\n", cudaStatus);
	}

	float millisecondsOp = 0;
	float millisecondsMem = 0;
	hipEventElapsedTime(&millisecondsOp, startOp, stopOp);
	hipEventElapsedTime(&millisecondsMem, startMem, stopMem);
	std::cout << "Spawn Boards Time:\n";
	PrintTiming(millisecondsOp, millisecondsMem);

	///******************************************BackTrack section*****************************************/

	//int * device_solved_board;
	//int * host_solved_board = (int *)malloc((BOARD_SIZE) * sizeof(int));
	//hipMalloc((void **)&device_solved_board, (BOARD_SIZE) * sizeof(int));

	//// start memory + solver timing
	//hipEventRecord(startMem);

	////Start timing math only
	//hipEventRecord(startOp);

	////Call Kernel
	//BackTracker << <dimGrid, dimBlock >> > (device_all_boards, device_solved_board); //Needs more variables

	////Stop puzzle only timing
	//hipEventRecord(stopOp);

	////Copy back to host
	//hipMemcpy(host_solved_board, device_solved_board, (BOARD_SIZE) * sizeof(int), hipMemcpyDeviceToHost);

	////Stop memory timing: sync must go here or it loses these timing events
	//hipEventRecord(stopMem);
	//cudaStatus = hipDeviceSynchronize();

	////Convert board back into Board class
	//for (int i = 0; i < BOARD_SIZE; i++)
	//{
	//	for (int j = 1; j < SUB_BOARD_SIZE + 1; j++)
	//	{
	//		_board->board[i][j] = false; //Set all values to false so the real true value will appear
	//	}
	//	_board->board[i][0] = true;
	//	_board->board[i][host_solved_board[i]] = true;
	//}

	free(all_boards);
	free(host_answer_board);
	hipFree(device_answer_board);
	return false;
}

int main()
{
	bool solved;
	int emptySpacesLeft;
	// Instantiates, Sets, and Prints out the initial game board
	Board *puzzle = SetBoard(test_board_medium);
	emptySpacesLeft = ReducePossibilities(puzzle);
	if (emptySpacesLeft == 0) //0 empty spaces
	{
		std::cout << "Puzzled solved using only crooks.\n";
		puzzle->print_board();
	}
	else
	{
		solved = BackTrack(puzzle, emptySpacesLeft);
		if (solved)
		{
			std::cout << "Puzzled solved using crooks + backtracking.\n";
			puzzle->print_board();
		}
	}

	
	puzzle->print_board(); // for testing

}
