#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <stdio.h>

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <set>
#include <chrono>
#include <string>


#define BOARD_SIZE 81
#define SUB_BOARD_SIZE 9
#define SUB_BOARD_DIM 3

#define THREAD_PER_BLOCK 1024

#pragma region Boards
/*Boards*/
// https://www.puzzles.ca/sudoku_puzzles/sudoku_easy_487.html
int* test_board_easy = new int[81]{ 0, 7, 0, 0, 0, 1, 0, 0, 0,
									6, 0, 0, 0, 0, 0, 0, 0, 0,
									0, 0, 0, 0, 5, 3, 0, 0, 0,
									0, 0, 0, 8, 0, 0, 0, 2, 0,
									0, 3, 0, 0, 4, 7, 1, 6, 0,
									4, 0, 0, 0, 0, 0, 5, 7, 0,
									0, 0, 0, 0, 1, 0, 7, 5, 0,
									0, 6, 0, 5, 2, 0, 0, 4, 0,
									3, 0, 0, 0, 0, 9, 0, 8, 6 };

int* easy_test_answer = new int[81]{ 5, 7, 9, 6, 8, 1, 4, 3, 2,
									6, 2, 3, 7, 9, 4, 8, 1, 5,
									1, 8, 4, 2, 5, 3, 6, 9, 7,
									7, 1, 6, 8, 3, 5, 9, 2, 4,
									2, 3, 5, 9, 4, 7, 1, 6, 8,
									4, 9, 8, 1, 6, 2, 5, 7, 3,
									8, 4, 2, 3, 1, 6, 7, 5, 9,
									9, 6, 7, 5, 2, 8, 3, 4, 1,
									3, 5, 1, 4, 7, 9, 2, 8, 6 };

int* test_board_easy2 = new int[81]{ 0, 0, 0, 0, 9, 0, 0, 0, 0,
									  0, 3, 0, 0, 0, 0, 7, 4, 2,
									  6, 1, 0, 0, 3, 0, 0, 0, 0,
									  0, 0, 8, 0, 0, 0, 0, 0, 0,
									  0, 2, 9, 1, 4, 7, 0, 0, 8,
									  4, 5, 0, 0, 0, 0, 0, 7, 0,
									  0, 4, 0, 0, 0, 0, 0, 6, 0,
									  5, 0, 0, 6, 0, 0, 0, 3, 0,
									  0, 0, 0, 0, 8, 1, 2, 0, 0 };

int* easy_test2_answer = new int[81]{ 2, 8, 4, 7, 9, 5, 3, 1, 6,
									   9, 3, 5, 8, 1, 6, 7, 4, 2,
									   6, 1, 7, 2, 3, 4, 5, 8, 9,
									   1, 7, 8, 5, 6, 9, 4, 2, 3,
									   3, 2, 9, 1, 4, 7, 6, 5, 8,
									   4, 5, 6, 3, 2, 8, 9, 7, 1,
									   8, 4, 2, 9, 5, 3, 1, 6, 7,
									   5, 9, 1, 6, 7, 2, 8, 3, 4,
									   7, 6, 3, 4, 8, 1, 2, 9, 5 };

// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_medium = new int[81]{ 0, 0, 0, 0, 0, 0, 3, 0, 0,
									   8, 5, 2, 3, 0, 0, 0, 0, 1,
									   0, 9, 0, 2, 0, 0, 0, 0, 4,
									   9, 7, 4, 0, 0, 0, 0, 0, 0,
									   0, 1, 0, 0, 6, 0, 0, 0, 0,
									   0, 0, 0, 0, 4, 0, 0, 0, 0,
									   6, 0, 9, 0, 8, 0, 0, 3, 7,
									   3, 0, 0, 0, 0, 0, 0, 6, 0,
									   0, 2, 0, 0, 0, 5, 0, 0, 0 };

int* medium_test_answer = new int[81]{ 4, 6, 7, 9, 1, 8, 3, 2, 5,
										8, 5, 2, 3, 7, 4, 6 ,9, 1,
										1, 9, 3, 2, 5, 6, 7, 8, 4,
										9, 7, 4, 5, 2, 3, 8, 1, 6,
										2, 1, 8, 7, 6, 9, 4, 5, 3,
										5, 3, 6, 8, 4, 1, 2, 7, 9,
										6, 4, 9, 1, 8, 2, 5, 3, 7,
										3, 8, 5, 4, 9, 7, 1, 6, 2,
										7, 2, 1, 6, 3, 5, 9, 4, 8 };


// https://www.puzzles.ca/sudoku_puzzles/sudoku_medium_487.html
int* test_board_hard = new int[81]{ 0, 7, 0, 5, 0, 6, 0, 0, 0,
									 4, 0, 3, 0, 0, 0, 0, 0, 1,
									 0, 6, 0, 0, 0, 0, 9, 0, 7,
									 0, 0, 0, 7, 3, 0, 8, 2, 0,
									 8, 0, 4, 0, 5, 0, 0, 7, 3,
									 0, 9, 0, 0, 2, 0, 0, 0, 5,
									 0, 0, 1, 0, 0, 0, 0, 0, 0,
									 0, 0, 0, 1, 0, 0, 2, 0, 6,
									 0, 0, 0, 3, 8, 2, 0, 0, 0 };

//int* hard_test_answer = new int[81]{    4, 6, 7, 9, 1, 8, 3, 2, 5,
//										8, 5, 2, 3, 7, 4, 6 ,9, 1,
//										1, 9, 3, 2, 5, 6, 7, 8, 4,
//										9, 7, 4, 5, 2, 3, 8, 1, 6,
//										2, 1, 8, 7, 6, 9, 4, 5, 3,
//										5, 3, 6, 8, 4, 1, 2, 7, 9,
//										6, 4, 9, 1, 8, 2, 5, 3, 7,
//										3, 8, 5, 4, 9, 7, 1, 6, 2,
//										7, 2, 1, 6, 3, 5, 9, 4, 8 };

// http://www.ams.org/notices/200904/rtx090400460p.pdf
int* test_board_diabolical = new int[81]{ 0, 9, 0, 7, 0, 0, 8, 6, 0,
										  0, 3, 1, 0, 0, 5, 0, 2, 0,
										  8, 0, 6, 0, 0, 0, 0, 0, 0,
										  0, 0, 7, 0, 5, 0, 0, 0, 6,
										  0, 0, 0, 3, 0, 7, 0, 0, 0,
										  5, 0, 0, 0, 1, 0, 7, 0, 0,
										  0, 0, 0, 0, 0, 0, 1, 0, 9,
										  0, 2, 0, 6, 0, 0, 3, 5, 0,
										  0, 5, 4, 0, 0, 8, 0, 7, 0 };

int* diabolical_test_answer = new int[81]{ 2, 9, 5, 7, 4, 3, 8, 6, 1,
										   4, 3, 1, 8, 6, 5, 9, 2, 7,
										   8, 7, 6, 1, 9, 2, 5, 4, 3,
										   3, 8, 7, 4, 5, 9, 2, 1, 6,
										   6, 1, 2, 3, 8, 7, 4, 9, 5,
										   5, 4, 9, 2, 1, 6, 7, 3, 8,
										   7, 6, 3, 5, 2, 4, 1, 8, 9,
										   9, 2, 8, 6, 7, 1, 3, 5, 4,
										   1, 5, 4, 9, 3, 8, 6, 7, 2 };

#pragma endregion


__device__ bool row_check_dev(const int* _board, int _board_root, int _row, int _entry, int loc, int _boardStart)
{
	for (int i = _row * _board_root; i < _row * _board_root + _board_root; i++) {
		if (i != loc && _board[i + _boardStart] == _entry)
		{
			return false;
		}
	}

	return true;
}

__device__ bool column_check_dev(const int* _board, int _board_root, int _col, int _entry, int loc, int _boardStart)
{
	for (int i = _col; i < _board_root * _board_root - (_board_root - _col); i += _board_root) {
		if (i != loc && _board[i + _boardStart] == _entry) {
			return false;
		}
	}

	return true;
}

__device__ bool grid_check_dev(const int* _board, int _board_root, int _start_row, int _start_col, int _entry, int loc, int _boardStart)
{
	int sub_grid_x = _start_row / SUB_BOARD_DIM; // 0, 1, or 2
	int sub_grid_y = _start_col / SUB_BOARD_DIM; // 0, 1, or 2
	int grid_start = (sub_grid_x * SUB_BOARD_SIZE * SUB_BOARD_DIM) + (sub_grid_y * SUB_BOARD_DIM);
	for (int i = 0; i < 3; i++) {
		for (int j = 0; j < 3; j++) {
			//		  start ind     rows of grid         col
			int ind = grid_start + (i * SUB_BOARD_SIZE) + j;
			if (ind != loc && _board[ind + _boardStart] == _entry) {
				return false;
			}
		}
	}

	return true;
}

__device__ bool is_legal_entry_dev(const int* _board, int _board_root, int _row, int _col, int _entry, int loc, int _boardStart)
{
	return row_check_dev(_board, _board_root, _row, _entry, loc, _boardStart) &&
		column_check_dev(_board, _board_root, _col, _entry, loc, _boardStart) &&
		grid_check_dev(_board, _board_root, _row, _col, _entry, loc, _boardStart);
}

// Returns whether or not it is valid to put a value in specified location for this board
__device__ bool IsLegal(int *_board, int _loc, int _val, int _boardStart)
{
	if (is_legal_entry_dev(_board, SUB_BOARD_SIZE, _loc / SUB_BOARD_DIM + _boardStart, _loc % SUB_BOARD_DIM + _boardStart, _val, _loc, _boardStart)) {
		_board[_loc] = _val;
		return true;
	}

	return false;
}

// Find next empty cell in passed in board
__device__ int FindNextEmptyCell(int* board, int _boardStart) {
	for (int i = 0; i < BOARD_SIZE; i++) 
	{
		if (board[i + _boardStart] == 0) {
			return i;
		}
	}
	return -1;
}

// new boards points to the end of the filled in prev boards
__global__ void GenerateBoardsByCell(int *old_boards, int old_board_num, int *new_boards, int *new_board_num) {

	// gives the previous board number to look at
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	// each thread will look at 1 previous board 
	// thread only does work if the amount of previous boards greater than its thread num
	// maybe should use a for loop in the case a thread has to do more than one thread. Will this ever occur?
	if (t_idx < old_board_num) {
		int old_board_start = t_idx * BOARD_SIZE;


		// find next index we can add to
		int empty_cell_ind = FindNextEmptyCell(old_boards, old_board_start);
		if (empty_cell_ind == -1) { // Board is full
			return;
		}

		// Now try all possible numbers in this cell that is legal
		for (int i = 1; i <= 9; i++) {
			if (IsLegal(old_boards, empty_cell_ind, i, old_board_start)) { // number can go in this cell

				// where to start appending for the new board
				int new_board_offset = atomicAdd(new_board_num, 1); // increment amount of boards at the new depth

				for (int j = 0; j < BOARD_SIZE; j++) 
				{
					int ind = (new_board_offset * BOARD_SIZE) + j;

					new_boards[ind] = old_boards[j + old_board_start];
				}
			}
		}
	}

}

// Use DFS to solve specified board per thread
__global__ void SolveBoard(int *boards, int total_boards, int* solution) {
	int t_idx = blockDim.x * blockIdx.x + threadIdx.x;

	// Each thread does DFS on 1 board
	if (t_idx < total_boards) 
	{
		int board_start = t_idx * BOARD_SIZE;
		int* thread_board = (int*)malloc(sizeof(int) * BOARD_SIZE);

		for (int i = 0; i < SUB_BOARD_SIZE; i++)// read prev board into a sudoku sized local array
		{ 
			thread_board[i] = boards[board_start + i];
		}



	}

}


// The main solve function to be called
void solve(int *board, int depth) {

	int h_solution[BOARD_SIZE];
	memset(h_solution, 0, BOARD_SIZE * sizeof(int));

	int *new_boards;
	int *old_boards;
	int *solution;
	int *next_board_num;

	const int memSize = 81 * pow(9, depth);

	// allocate device memory and set everything to 0
	hipMalloc(&next_board_num, sizeof(int));
	hipMalloc(&solution, BOARD_SIZE * sizeof(int));
	hipMalloc(&new_boards, memSize * sizeof(int));
	hipMalloc(&old_boards, memSize * sizeof(int));

	hipMemset(next_board_num, 0, sizeof(int));
	hipMemset(solution, 0, BOARD_SIZE * sizeof(int));
	hipMemset(new_boards, 0, memSize * sizeof(int));
	hipMemset(old_boards, 0, memSize * sizeof(int));


	// BFS on GPU

	// copy starting board to device memory
	hipMemcpy(old_boards, board, BOARD_SIZE * sizeof(int), hipMemcpyHostToDevice);

	// 1 due to starting board
	int old_board_num = 1;
	for (int i = 0; i < depth; i++) {

		// Need 1 thread per old board
		int num_blocks = (old_board_num + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

		// need to set next board num to 0 before we start next depth
		hipMemset(next_board_num, 0, sizeof(int));

		// we will read from old boards first and put into new boards
		GenerateBoardsByCell << <num_blocks, THREAD_PER_BLOCK >> > (old_boards, old_board_num, new_boards, next_board_num);

		// old and new boards are swapped in order for us to reuse the memory since we only care about the last depth
		
		int* temp_board = old_boards;
		old_boards = new_boards;
		new_boards = temp_board;

		// update the amount of boards in previous depth for next iteration
		hipMemcpy(&old_board_num, next_board_num, sizeof(int), hipMemcpyDeviceToHost);
	}


	// DFS on GPU
	// total amount of boards is just the boards in last depth
	int total_board_num = old_board_num;
	//hipMemcpy(&total_board_num, next_board_num, sizeof(int), hipMemcpyDeviceToHost);

	int num_blocks = (total_board_num + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK;

	SolveBoard << <num_blocks, THREAD_PER_BLOCK >> > (old_boards, total_board_num, solution);

	hipDeviceSynchronize();

	// copy the devicce solution to the host
	hipMemcpy(h_solution, solution, BOARD_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	// Print board?

	hipFree(new_boards);
	hipFree(&old_boards);
	hipFree(&solution);
	hipFree(&next_board_num);

	return;
}

int main()
{
	// Solve medium sized board and go to depth 7 of BFS
	//solve(test_board_medium, 7);
	solve(test_board_medium, 1);
}
